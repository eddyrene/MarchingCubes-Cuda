
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string>
#include <fstream>
#include <cmath>
#include <string>
#include <iostream>

using namespace std;

typedef struct {
   double x,y,z;
} XYZ;

typedef struct {
   XYZ p[8];
   double val[8];
} GRIDCELL;

typedef struct {
   XYZ p[3];         /* Vertices */
   XYZ c;            /* Centroid */
   XYZ n[3];         /* Normal   */
} TRIANGLE;

#define ABS(x) (x < 0 ? -(x) : (x))

// Prototypes
//__global__
//int PolygoniseCube(GRIDCELL,double,TRIANGLE *);
//XYZ VertexInterp(double,XYZ,XYZ,double,double);
/*
#define NX 200
#define NY 160
#define NZ 160
*/
int NX = 100;
int NY = 160;
int NZ = 160;

void fillMatrix(XYZ* a, int n)
{
   int i;
   for (i = 0; i < n; ++i)
   {
        a[i].x = 3;
        a[i].y = 2;
        a[i].z = 5;//rand()%5;
   }
}
__global__
void matrixAdition(XYZ * b, XYZ *a,int n)
{
   	int ij = threadIdx.x + blockDim.x * blockIdx.x;

	if(ij<n)
	{
		b[ij].x = a[ij].x+2;
		b[ij].y = a[ij].y+3;
		b[ij].z = a[ij].z+0;
		//printf("da %d \n" , b[ij].x);
	}
}
void printMatrix(string s, XYZ *a , int tam){
	cout<<s;
	for(int i=0;i<tam;i++)
	{
		cout<<a[i].x<<" "<<a[i].y<<" "<<a[i].z<<" ";
		cout<<endl;
	}
}
void assingMem(int *** data)
{
	int i,j;
	data = (int ***)malloc(NX*sizeof(short int **));
	for (i=0;i<NX;i++)
		data[i] = (int **)malloc(NY*sizeof(short int *));
		for (i=0;i<NX;i++)
			for (j=0;j<NY;j++)
				data[i][j] = (int *)malloc(NZ*sizeof(short int));

}
void readFile(FILE *fptr, const char * namefile , int themin , int themax, int *** data)
{
	int i,j,k,c;
	fprintf(stderr,"Load data ...\n");
	if ((fptr = fopen(namefile,"rb")) == NULL) {
		fprintf(stderr,"Error al leer archivo\n");
		exit(-1);
	}
	for (k=0;k<NZ;k++) {
		for (j=0;j<NY;j++) {
			for (i=0;i<NX;i++) {
				if ((c = fgetc(fptr)) == EOF) {
					fprintf(stderr,"Error en tamaño\n");
					exit(-1);
				}
				data[i][j][k] = c;
				cout<<"leyendo :"<<c<<endl;
				if (c > themax)
					themax = c;
				if (c < themin)
					themin = c;
			}
		}
	}
	fclose(fptr);
	fprintf(stderr,"Rango del volumen: %d -> %d\n",themin,themax);
}

void constructCubes(GRIDCELL * vectGrids, int *** data, int gtam)
{
		int i,j,k;
		//fprintf(stderr,"Construyendo Cubos ...\n");
		int cont=0;
		for (i=0;i<NX-1;i++) {
			//cout<<i<<endl;
			//if (i % (NX/10) == 0)
				//fprintf(stderr,"   Slice %d de %d\n",i,NX);
			for (j=0;j<NY-1;j++) {
				for (k=0;k<NZ-1;k++) {
					GRIDCELL grid;
					grid.p[0].x = i;
					grid.p[0].y = j;
		         	grid.p[0].z = k;
						grid.val[0] = data[i][j][k];
		            grid.p[1].x = i+1;
		            grid.p[1].y = j;
		            grid.p[1].z = k;
						grid.val[1] = data[i+1][j][k];
		            grid.p[2].x = i+1;
		            grid.p[2].y = j+1;
		            grid.p[2].z = k;
						grid.val[2] = data[i+1][j+1][k];
		            grid.p[3].x = i;
		            grid.p[3].y = j+1;
		            grid.p[3].z = k;
						grid.val[3] = data[i][j+1][k];
		            grid.p[4].x = i;
		            grid.p[4].y = j;
		            grid.p[4].z = k+1;
						grid.val[4] = data[i][j][k+1];
		            grid.p[5].x = i+1;
		            grid.p[5].y = j;
		            grid.p[5].z = k+1;
						grid.val[5] = data[i+1][j][k+1];
		            grid.p[6].x = i+1;
		            grid.p[6].y = j+1;
		            grid.p[6].z = k+1;
						grid.val[6] = data[i+1][j+1][k+1];
		            grid.p[7].x = i;
		            grid.p[7].y = j+1;
		            grid.p[7].z = k+1;
						grid.val[7] = data[i][j+1][k+1];
					vectGrids[i+j*NY+k*NY*NZ]=grid;
					cont++;
					//cout<<cont<<endl;
				}
			}
		}

}
__device__
XYZ VertexInterp(double isolevel,XYZ p1,XYZ p2,double valp1,double valp2)
{
   double mu;
   XYZ p;
   if (ABS(isolevel-valp1) < 0.00001)
      return(p1);
   if (ABS(isolevel-valp2) < 0.00001)
      return(p2);
   if (ABS(valp1-valp2) < 0.00001)
      return(p1);
   mu = (isolevel - valp1) / (valp2 - valp1);
   p.x = p1.x + mu * (p2.x - p1.x);
   p.y = p1.y + mu * (p2.y - p1.y);
   p.z = p1.z + mu * (p2.z - p1.z);
   return p;
}

__device__
void copyXYZ(XYZ &a, XYZ &b)
{
	a.x=b.x ; a.y=b.y ; a.z = b.z;
}

__device__
XYZ defect()
{
	XYZ a; 
	a.x=300 ; a.y=300 ; a.z = 300;
	return a;
}


__global__
void coyGRID(GRIDCELL * a, GRIDCELL * b, int x, int y, int z)
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
    int j = threadIdx.y + blockDim.y * blockIdx.y;
    int k = threadIdx.z + blockDim.z * blockIdx.z;
	
	/*if(i<x && j<y && k<z)
	{
		a[ij].p = b[ij].p;
		a[ij].val = b[ij].val;
	}*/
}

__global__
void copyGRID1(GRIDCELL * a, GRIDCELL * b, int x, int y, int z)
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
    int j = threadIdx.y + blockDim.y * blockIdx.y;
    int k = threadIdx.z + blockDim.z * blockIdx.z;
	
	if(i<x && j<y && k<z)
	{
		for(int w=0;w<8;w++)
		{
			a[i+j*y+k*y*z].p[w] = b[i+j*y+k*y*z].p[w];
			a[i+j*y+k*y*z].val[w] = b[i+j*y+k*y*z].val[w];
		}
	}
}

/*
__global__
void PolygoniseCube(XYZ * vertlist ,GRIDCELL * g ,double iso, int x ,int y , int z)
*/

__global__
void PolygoniseCube(XYZ * vertlist ,GRIDCELL * g ,double iso, int x ,int y , int z)
{
	//printf("g %d \n",iso);
	int i = threadIdx.x + blockDim.x * blockIdx.x;
    int j = threadIdx.y + blockDim.y * blockIdx.y;
    int k = threadIdx.z + blockDim.z * blockIdx.z;
	if(i<x && j<y && k<z)
	{

		//printf("thread %d \n", g[i].p[7].x);
		int cubeindex;
		//int tamVert=12;
		//XYZ vertlist[12];
		int edgeTable[256]={
		0x0  , 0x109, 0x203, 0x30a, 0x406, 0x50f, 0x605, 0x70c,
		0x80c, 0x905, 0xa0f, 0xb06, 0xc0a, 0xd03, 0xe09, 0xf00,
		0x190, 0x99 , 0x393, 0x29a, 0x596, 0x49f, 0x795, 0x69c,
		0x99c, 0x895, 0xb9f, 0xa96, 0xd9a, 0xc93, 0xf99, 0xe90,
		0x230, 0x339, 0x33 , 0x13a, 0x636, 0x73f, 0x435, 0x53c,
		0xa3c, 0xb35, 0x83f, 0x936, 0xe3a, 0xf33, 0xc39, 0xd30,
		0x3a0, 0x2a9, 0x1a3, 0xaa , 0x7a6, 0x6af, 0x5a5, 0x4ac,
		0xbac, 0xaa5, 0x9af, 0x8a6, 0xfaa, 0xea3, 0xda9, 0xca0,
		0x460, 0x569, 0x663, 0x76a, 0x66 , 0x16f, 0x265, 0x36c,
		0xc6c, 0xd65, 0xe6f, 0xf66, 0x86a, 0x963, 0xa69, 0xb60,
		0x5f0, 0x4f9, 0x7f3, 0x6fa, 0x1f6, 0xff , 0x3f5, 0x2fc,
		0xdfc, 0xcf5, 0xfff, 0xef6, 0x9fa, 0x8f3, 0xbf9, 0xaf0,
		0x650, 0x759, 0x453, 0x55a, 0x256, 0x35f, 0x55 , 0x15c,
		0xe5c, 0xf55, 0xc5f, 0xd56, 0xa5a, 0xb53, 0x859, 0x950,
		0x7c0, 0x6c9, 0x5c3, 0x4ca, 0x3c6, 0x2cf, 0x1c5, 0xcc ,
		0xfcc, 0xec5, 0xdcf, 0xcc6, 0xbca, 0xac3, 0x9c9, 0x8c0,
		0x8c0, 0x9c9, 0xac3, 0xbca, 0xcc6, 0xdcf, 0xec5, 0xfcc,
		0xcc , 0x1c5, 0x2cf, 0x3c6, 0x4ca, 0x5c3, 0x6c9, 0x7c0,
		0x950, 0x859, 0xb53, 0xa5a, 0xd56, 0xc5f, 0xf55, 0xe5c,
		0x15c, 0x55 , 0x35f, 0x256, 0x55a, 0x453, 0x759, 0x650,
		0xaf0, 0xbf9, 0x8f3, 0x9fa, 0xef6, 0xfff, 0xcf5, 0xdfc,
		0x2fc, 0x3f5, 0xff , 0x1f6, 0x6fa, 0x7f3, 0x4f9, 0x5f0,
		0xb60, 0xa69, 0x963, 0x86a, 0xf66, 0xe6f, 0xd65, 0xc6c,
		0x36c, 0x265, 0x16f, 0x66 , 0x76a, 0x663, 0x569, 0x460,
		0xca0, 0xda9, 0xea3, 0xfaa, 0x8a6, 0x9af, 0xaa5, 0xbac,
		0x4ac, 0x5a5, 0x6af, 0x7a6, 0xaa , 0x1a3, 0x2a9, 0x3a0,
		0xd30, 0xc39, 0xf33, 0xe3a, 0x936, 0x83f, 0xb35, 0xa3c,
		0x53c, 0x435, 0x73f, 0x636, 0x13a, 0x33 , 0x339, 0x230,
		0xe90, 0xf99, 0xc93, 0xd9a, 0xa96, 0xb9f, 0x895, 0x99c,
		0x69c, 0x795, 0x49f, 0x596, 0x29a, 0x393, 0x99 , 0x190,
		0xf00, 0xe09, 0xd03, 0xc0a, 0xb06, 0xa0f, 0x905, 0x80c,
		0x70c, 0x605, 0x50f, 0x406, 0x30a, 0x203, 0x109, 0x0   };


		//int i,ntri = 0;

		cubeindex = 0;

		if (g[i+j*y+k*y*z].val[0] < iso) cubeindex |= 1;
		if (g[i+j*y+k*y*z].val[1] < iso) cubeindex |= 2;
		if (g[i+j*y+k*y*z].val[2] < iso) cubeindex |= 4;
		if (g[i+j*y+k*y*z].val[3] < iso) cubeindex |= 8;
		if (g[i+j*y+k*y*z].val[4] < iso) cubeindex |= 16;
		if (g[i+j*y+k*y*z].val[5] < iso) cubeindex |= 32;
		if (g[i+j*y+k*y*z].val[6] < iso) cubeindex |= 64;
		if (g[i+j*y+k*y*z].val[7] < iso) cubeindex |= 128;
		

		//XYZ a;
		//a.x=20 ; a.y=50; a,z=0;
		//vertlist[i+j*y+k*y*z+0].x=g[i+j*y+k*y*z].val[6];
		//vertlist[i+j*y+k*y*z+0].y=10;
		//vertlist[i+j*y+k*y*z+0].z=10;*/
	   /* Cube is entirely in/out of the surface */
	   if (edgeTable[cubeindex] == 0)
	      return;
	   /* Find the vertices where the surface intersects the cube */
	   if (edgeTable[cubeindex] & 1) {
	      vertlist[i+j*y+k*y*z+0] = VertexInterp(iso,g[i+j*y+k*y*z].p[0],g[i+j*y+k*y*z].p[1],g[i+j*y+k*y*z].val[0],g[i+j*y+k*y*z].val[1]);
	   }
	   if (edgeTable[cubeindex] & 2) {
	      vertlist[i+j*y+k*y*z+1] = VertexInterp(iso,g[i+j*y+k*y*z].p[1],g[i+j*y+k*y*z].p[2],g[i+j*y+k*y*z].val[1],g[i+j*y+k*y*z].val[2]);
	   }
	   if (edgeTable[cubeindex] & 4) {
	      vertlist[i+j*y+k*y*z+2] = VertexInterp(iso,g[i+j*y+k*y*z].p[2],g[i+j*y+k*y*z].p[3],g[i+j*y+k*y*z].val[2],g[i+j*y+k*y*z].val[3]);
	   }
	   if (edgeTable[cubeindex] & 8) {
	      vertlist[i+j*y+k*y*z+3] = VertexInterp(iso,g[i+j*y+k*y*z].p[3],g[i+j*y+k*y*z].p[0],g[i+j*y+k*y*z].val[3],g[i+j*y+k*y*z].val[0]);
	   }
	   if (edgeTable[cubeindex] & 16) {
	      vertlist[i+j*y+k*y*z+4] = VertexInterp(iso,g[i+j*y+k*y*z].p[4],g[i+j*y+k*y*z].p[5],g[i+j*y+k*y*z].val[4],g[i+j*y+k*y*z].val[5]);
	   }
	   if (edgeTable[cubeindex] & 32) {
	      vertlist[i+j*y+k*y*z+5] = VertexInterp(iso,g[i+j*y+k*y*z].p[5],g[i+j*y+k*y*z].p[6],g[i+j*y+k*y*z].val[5],g[i+j*y+k*y*z].val[6]);
	   }
	   if (edgeTable[cubeindex] & 64) {
	      vertlist[i+j*y+k*y*z+6] = VertexInterp(iso,g[i+j*y+k*y*z].p[6],g[i+j*y+k*y*z].p[7],g[i+j*y+k*y*z].val[6],g[i+j*y+k*y*z].val[7]);
	   }
	   if (edgeTable[cubeindex] & 128) {
	      vertlist[i+j*y+k*y*z+7] = VertexInterp(iso,g[i+j*y+k*y*z].p[7],g[i+j*y+k*y*z].p[4],g[i+j*y+k*y*z].val[7],g[i+j*y+k*y*z].val[4]);
	   }
	   if (edgeTable[cubeindex] & 256) {
	      vertlist[i+j*y+k*y*z+8] = VertexInterp(iso,g[i+j*y+k*y*z].p[0],g[i+j*y+k*y*z].p[4],g[i+j*y+k*y*z].val[0],g[i+j*y+k*y*z].val[4]);
	   }
	   if (edgeTable[cubeindex] & 512) {
	      vertlist[i+j*y+k*y*z+9] = VertexInterp(iso,g[i+j*y+k*y*z].p[1],g[i+j*y+k*y*z].p[5],g[i+j*y+k*y*z].val[1],g[i+j*y+k*y*z].val[5]);
	   }
	   if (edgeTable[cubeindex] & 1024) {
	      vertlist[i+j*y+k*y*z+10] = VertexInterp(iso,g[i+j*y+k*y*z].p[2],g[i+j*y+k*y*z].p[6],g[i+j*y+k*y*z].val[2],g[i+j*y+k*y*z].val[6]);
	   }
	   if (edgeTable[cubeindex] & 2048) {
	      vertlist[i+j*y+k*y*z+11] = VertexInterp(iso,g[i+j*y+k*y*z].p[3],g[i+j*y+k*y*z].p[7],g[i+j*y+k*y*z].val[3],g[i+j*y+k*y*z].val[7]);
	   }
	  // printf("hasta aqui llega \n");
	   
	}
}


void printGrid(string a, GRIDCELL * g, int tam)
{
	cout<<a;
	for(int i =0; i<tam ;i++)
		for(int j=0;j<8;j++)
			//printf("%f  %f  %f \n", g[i].p[j].x ,g[i].p[j].y,g[i].p[j].z);
		      printf("%f \n", g[i].val[j]);		
}

int main(int argc, char *argv[])
{
	int i,j,k,c;
	int ***data;
	FILE *fptr;
	int N= (NX*NY*NZ);
	cout<<N<<endl; //return 1;
	int THREADS_PER_BLOCK =8;
	int themin=255;
	int themax=0;
	int isolevel=80;
	//const char* FILENAME = "mri.raw";
	//assingMem(data);
	//readFile(fptr,FILENAME,themin, themax,data);

	// Malloc the volumetric data, hardwired size!
	data = (int***)malloc(NX*sizeof(int **));
	for (i=0;i<NX;i++)
		data[i] = (int**)malloc(NY*sizeof(int *));
	for (i=0;i<NX;i++)
		for (j=0;j<NY;j++)
			data[i][j] = (int*)malloc(NZ*sizeof(int));

	//cout<<data[199][60][0]<<endl;
	// Open and read the raw data
	fprintf(stderr,"Reading data ...\n");
	if ((fptr = fopen(argv[argc-1],"rb")) == NULL) {
		fprintf(stderr,"File open failed\n");
		exit(-1);
	}
	cout<<"llega"<<endl;
	for (k=0;k<NZ;k++) {
		for (j=0;j<NY;j++) {
			for (i=0;i<NX;i++) {
				if ((c = fgetc(fptr)) == EOF) {
					fprintf(stderr,"Unexpected end of file\n");
					exit(-1);
				}
				data[i][j][k] = c;
				//cout<<i<<" "<<j <<" "<<k <<" data : "<<data[i][j][k]<<endl;
				if (c > themax)
					themax = c;
				if (c < themin)
					themin = c;
			}
		}
	}
	fclose(fptr);
	fprintf(stderr,"Volumetric data range: %d -> %d\n",themin,themax);

	int sizeGRID = N*sizeof(GRIDCELL);
	cout<<"pasa"<<endl;
	int sizeXYZ  = N*12*sizeof(XYZ);

	cout<<"sizeGRID "<<sizeGRID<<endl;
	cout<<"sizeXYZ "<<sizeXYZ<<endl;
	
	//cudaMalloc((void **)&d_b, size);

	GRIDCELL * vectGrids;
	GRIDCELL * d_vectGrid;
	XYZ * d_points;
	XYZ * points;
	points = (XYZ *)malloc(sizeXYZ);
	vectGrids = (GRIDCELL *)malloc(sizeGRID);
	constructCubes(vectGrids,data,N);
	/*
		typedef struct {
		double x,y,z;
		} XYZ;

		typedef struct {
		XYZ p[8];
		double val[8];
		} GRIDCELL;	
	*/	
	XYZ * d_p; double * d_val;

	size_t available, total;
	hipMemGetInfo(&available, &total);
	cout<<"available:  " << available<<" total:  "<<total <<endl;
	hipMalloc((void **)&d_vectGrid, sizeGRID);

	/*
	for(int i=0;i<N;i++)
	{
		cudaMalloc((void**)&d_p,8*sizeof(XYZ));
		//cudaMemGetInfo(&available, &total);
		//cout<<"available:  " << available<<" total:  "<<total <<endl;
		cudaMalloc((void**)&d_val,8*sizeof(double));
		//cudaMemGetInfo(&available, &total);
		//cout<<"available:  " << available<<" total:  "<<total <<endl;
		cudaMemcpy(d_p,vectGrids[i].p,8*sizeof(XYZ),cudaMemcpyHostToDevice);
		//for(int w=0;w<8;w++)
		//{
			cout<<vectGrids[i].p[w].y<<endl;
		//}
		//cudaMemGetInfo(&available, &total);
		//cout<<"available:  " << available<<" total:  "<<total <<endl;
		cudaMemcpy(d_val,vectGrids[i].val,8*sizeof(double),cudaMemcpyHostToDevice);
		//cudaMemGetInfo(&available, &total);
		//cout<<"available:  " << available<<" total:  "<<total <<endl
		cudaMemGetInfo(&available, &total);
		//cout<<"available:  " << available<<" total:  "<<total <<endl;
		cudaMemcpy(d_vectGrid[i].val, d_val, 8*sizeof(double),cudaMemcpyHostToDevice);
		cudaMemcpy(d_vectGrid[i].p, d_p, 8*sizeof(XYZ),cudaMemcpyHostToDevice);
 	 }*/
	hipMemcpy(d_vectGrid,vectGrids, sizeGRID, hipMemcpyHostToDevice);	
	cout<<"termino de asignar memoria"<<endl;
 	 XYZ * d_a, * d_sal;
 	GRIDCELL * d_res;
 	d_sal=(XYZ *)malloc(sizeXYZ);
 	hipMalloc((void **)&d_res, sizeGRID);
 	hipMalloc((void **)&d_a, sizeXYZ);
	hipMalloc((void **)&d_points, sizeXYZ);
	//cout<<"grid  "<<vectGrids<<endl;
	//cout<<"point "<<points<<endl;

	//fillMatrix(points, N);
	printMatrix("imprimiendo pruevba",points, 10);

	hipMemcpy(d_points, points, sizeXYZ, hipMemcpyHostToDevice);
	
	cout<<"grid "<<d_vectGrid<<endl;
	cout<<"pointsssss "<<d_points<<endl;
	//printf("dir %d \n",*d_points);
	cout<<"separa memoria sin problemas"<<endl;
	//printGrid("imprimiendo Grid inicial en Host \n ",vectGrids,N);
	hipEvent_t start, stop;
	float elapsedTime;
	hipEventCreate(&start);


	int x = NX; int y = NY ; int z = NZ;
	int blockX= (NX + THREADS_PER_BLOCK -1)/THREADS_PER_BLOCK;
	int blockY= (NY + THREADS_PER_BLOCK -1)/THREADS_PER_BLOCK;
	int blockZ= (NZ + THREADS_PER_BLOCK -1)/THREADS_PER_BLOCK;
	cout<<"blocks : "<<blockX<<" threds:  "<<THREADS_PER_BLOCK<<endl;
	cout<<"blocks : "<<blockY<<" threds:  "<<THREADS_PER_BLOCK<<endl;
	cout<<"blocks : "<<blockZ<<" threds:  "<<THREADS_PER_BLOCK<<endl;
	//int blocks= (10 + THREADS_PER_BLOCK -1)/THREADS_PER_BLOCK;
	/*cout<<"blocks : \n"<<blocks<<"\n threds: \n "<<THREADS_PER_BLOCK<<endl; */

	dim3 dimGrid(blockX, blockY, blockZ);
	dim3 dimBlock(THREADS_PER_BLOCK,THREADS_PER_BLOCK, THREADS_PER_BLOCK);
	hipEventRecord(start,0);
	isolevel=10;

		//copyGRID1<<<dimGrid,dimBlock>>>(d_res,d_vectGrid,x,y,z);
		PolygoniseCube<<<dimGrid,dimBlock>>>(d_points,d_vectGrid,isolevel,x,y,z);
		//PolygoniseCube<<<blocks,THREADS_PER_BLOCK>>>(d_points,d_vectGrids,isolevel);
		//matrixAdition<<<blocks,THREADS_PER_BLOCK>>>(d_a, d_points,10);
		//matrixAditionCol<<<blocks2,THREADS_PER_BLOCK>>>( d_c, d_a, d_b,N);
	hipEventCreate(&stop);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start,stop);
	printf("Elapsed time : %f ms\n" ,elapsedTime);
	hipMemcpy(points,d_points, sizeXYZ, hipMemcpyDeviceToHost);
	//GRIDCELL * res;
	//res = (GRIDCELL *)malloc(sizeGRID);
	//cudaMemcpy(res,d_vectGrid, sizeGRID, cudaMemcpyDeviceToHost);
	//printGrid("imprimiendo Grid final despues de la copia \n ",res,N);

	printMatrix("Printing Matrix  A \n",points,N);
	/*/printMatrix("Printing Matrix B \n",b,N);
	//printMatrix("Printing Matrix C \n",c,N);
	*/
	free(points); free(vectGrids);
	hipFree(d_points); hipFree(d_vectGrid);
	return 0;
}
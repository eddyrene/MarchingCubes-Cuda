
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string>
#include <fstream>
#include <cmath>
#include <string>
#include <iostream>

using namespace std;

typedef struct {
   double x,y,z;
} XYZ;

typedef struct {
   XYZ p[8];
   double val[8];
} GRIDCELL;

typedef struct {
   XYZ p[3];         /* Vertices */
 //  XYZ c;            /* Centroid */
 //  XYZ n[3];         /* Normal   */
} TRIANGLE;

#define ABS(x) (x < 0 ? -(x) : (x))

// Prototypes
//__global__
//int PolygoniseCube(GRIDCELL,double,TRIANGLE *);
//XYZ VertexInterp(double,XYZ,XYZ,double,double);
/*
#define NX 200
#define NY 160
#define NZ 160
*/
#define NX 68//200
#define NY 128//160
#define NZ 128//160

//68 x 256 x 256

void fillMatrix(XYZ* a, int n)
{
   int i;
   for (i = 0; i < n; ++i)
   {
        a[i].x = 3;
        a[i].y = 2;
        a[i].z = 5;//rand()%5;
   }
}
__global__
void matrixAdition(XYZ * b, XYZ *a,int n)
{
   	int ij = threadIdx.x + blockDim.x * blockIdx.x;

	if(ij<n)
	{
		b[ij].x = a[ij].x+2;
		b[ij].y = a[ij].y+3;
		b[ij].z = a[ij].z+0;
		//printf("da %d \n" , b[ij].x);
	}
}
void printMatrix(string s, XYZ *a , int tam){
	cout<<s;
	for(int i=0;i<tam;i++)
	{
		if(a[i].x!=0 && a[i].y!=0 && a[i].z!=0)
		{
			cout<<a[i].x<<" "<<a[i].y<<" "<<a[i].z<<" ";
			cout<<endl;
		}
	}
}
void assingMem(int *** data)
{
	int i,j;
	data = (int ***)malloc(NX*sizeof(short int **));
	for (i=0;i<NX;i++)
		data[i] = (int **)malloc(NY*sizeof(short int *));
		for (i=0;i<NX;i++)
			for (j=0;j<NY;j++)
				data[i][j] = (int *)malloc(NZ*sizeof(short int));

}
void readFile(FILE *fptr, const char * namefile , int themin , int themax, int *** data)
{
	int i,j,k,c;
	fprintf(stderr,"Load data ...\n");
	if ((fptr = fopen(namefile,"rb")) == NULL) {
		fprintf(stderr,"Error al leer archivo\n");
		exit(-1);
	}
	for (k=0;k<NZ;k++) {
		for (j=0;j<NY;j++) {
			for (i=0;i<NX;i++) {
				if ((c = fgetc(fptr)) == EOF) {
					fprintf(stderr,"Error en tamaño\n");
					exit(-1);
				}
				data[i][j][k] = c;
				cout<<"leyendo :"<<c<<endl;
				if (c > themax)
					themax = c;
				if (c < themin)
					themin = c;
			}
		}
	}
	fclose(fptr);
	fprintf(stderr,"Rango del volumen: %d -> %d\n",themin,themax);
}

void constructCubes(GRIDCELL * vectGrids, int *** data, int gtam)
{
		int i,j,k;
		//fprintf(stderr,"Construyendo Cubos ...\n");
		int cont=0;
		for (i=0;i<NX-1;i++) {
			//cout<<i<<endl;
			//if (i % (NX/10) == 0)
				//fprintf(stderr,"   Slice %d de %d\n",i,NX);
			for (j=0;j<NY-1;j++) {
				for (k=0;k<NZ-1;k++) {
					GRIDCELL grid;
					grid.p[0].x = i;
					grid.p[0].y = j;
		         	grid.p[0].z = k;
						grid.val[0] = data[i][j][k];
		            grid.p[1].x = i+1;
		            grid.p[1].y = j;
		            grid.p[1].z = k;
						grid.val[1] = data[i+1][j][k];
		            grid.p[2].x = i+1;
		            grid.p[2].y = j+1;
		            grid.p[2].z = k;
						grid.val[2] = data[i+1][j+1][k];
		            grid.p[3].x = i;
		            grid.p[3].y = j+1;
		            grid.p[3].z = k;
						grid.val[3] = data[i][j+1][k];
		            grid.p[4].x = i;
		            grid.p[4].y = j;
		            grid.p[4].z = k+1;
						grid.val[4] = data[i][j][k+1];
		            grid.p[5].x = i+1;
		            grid.p[5].y = j;
		            grid.p[5].z = k+1;
						grid.val[5] = data[i+1][j][k+1];
		            grid.p[6].x = i+1;
		            grid.p[6].y = j+1;
		            grid.p[6].z = k+1;
						grid.val[6] = data[i+1][j+1][k+1];
		            grid.p[7].x = i;
		            grid.p[7].y = j+1;
		            grid.p[7].z = k+1;
						grid.val[7] = data[i][j+1][k+1];
					
					vectGrids[i+j*NY+k*NY*NZ]=grid;
					cont++;
					//cout<<cont<<endl;
				}
			}
		}
}
__device__
XYZ VertexInterp(double isolevel,XYZ p1,XYZ p2,double valp1,double valp2)
{
   double mu;
   XYZ p;
   if (ABS(isolevel-valp1) < 0.00001)
      return(p1);
   if (ABS(isolevel-valp2) < 0.00001)
      return(p2);
   if (ABS(valp1-valp2) < 0.00001)
      return(p1);
   mu = (isolevel - valp1) / (valp2 - valp1);
   p.x = p1.x + mu * (p2.x - p1.x);
   p.y = p1.y + mu * (p2.y - p1.y);
   p.z = p1.z + mu * (p2.z - p1.z);
   return p;
}

__device__
void copyXYZ(XYZ &a, XYZ &b)
{
	a.x=b.x ; a.y=b.y ; a.z = b.z;
}

__device__
XYZ defect()
{
	XYZ a; 
	a.x=300 ; a.y=300 ; a.z = 300;
	return a;
}


__global__
void coyGRID(GRIDCELL * a, GRIDCELL * b, int x, int y, int z)
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
    int j = threadIdx.y + blockDim.y * blockIdx.y;
    int k = threadIdx.z + blockDim.z * blockIdx.z;
	/*if(i<x && j<y && k<z)
	{
		a[ij].p = b[ij].p;
		a[ij].val = b[ij].val;
	}*/
}
__global__
void copyGRID1(GRIDCELL * a, GRIDCELL * b, int x, int y, int z)
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
    int j = threadIdx.y + blockDim.y * blockIdx.y;
    int k = threadIdx.z + blockDim.z * blockIdx.z;
	
	if(i<x && j<y && k<z)
	{
		for(int w=0;w<8;w++)
		{
			a[i+j*y+k*y*z].p[w] = b[i+j*y+k*y*z].p[w];
			a[i+j*y+k*y*z].val[w] = b[i+j*y+k*y*z].val[w];
		}
	}
}
/*
__global__
void PolygoniseCube(XYZ * vertlist ,GRIDCELL * g ,double iso, int x ,int y , int z)
*/
__global__
void PolygoniseCube(TRIANGLE * tris,GRIDCELL * g ,double iso, int x ,int y , int z)
{
	//printf("g %d \n",iso);
	int i = threadIdx.x + blockDim.x * blockIdx.x;
    int j = threadIdx.y + blockDim.y * blockIdx.y;
    int k = threadIdx.z + blockDim.z * blockIdx.z;
	if(i<x && j<y && k<z)
	{
		//printf("thread %d \n", g[i].p[7].x);
		int cubeindex;
		//int tamVert=12;
		XYZ vertlist[12];
		int edgeTable[256]={
		0x0  , 0x109, 0x203, 0x30a, 0x406, 0x50f, 0x605, 0x70c,
		0x80c, 0x905, 0xa0f, 0xb06, 0xc0a, 0xd03, 0xe09, 0xf00,
		0x190, 0x99 , 0x393, 0x29a, 0x596, 0x49f, 0x795, 0x69c,
		0x99c, 0x895, 0xb9f, 0xa96, 0xd9a, 0xc93, 0xf99, 0xe90,
		0x230, 0x339, 0x33 , 0x13a, 0x636, 0x73f, 0x435, 0x53c,
		0xa3c, 0xb35, 0x83f, 0x936, 0xe3a, 0xf33, 0xc39, 0xd30,
		0x3a0, 0x2a9, 0x1a3, 0xaa , 0x7a6, 0x6af, 0x5a5, 0x4ac,
		0xbac, 0xaa5, 0x9af, 0x8a6, 0xfaa, 0xea3, 0xda9, 0xca0,
		0x460, 0x569, 0x663, 0x76a, 0x66 , 0x16f, 0x265, 0x36c,
		0xc6c, 0xd65, 0xe6f, 0xf66, 0x86a, 0x963, 0xa69, 0xb60,
		0x5f0, 0x4f9, 0x7f3, 0x6fa, 0x1f6, 0xff , 0x3f5, 0x2fc,
		0xdfc, 0xcf5, 0xfff, 0xef6, 0x9fa, 0x8f3, 0xbf9, 0xaf0,
		0x650, 0x759, 0x453, 0x55a, 0x256, 0x35f, 0x55 , 0x15c,
		0xe5c, 0xf55, 0xc5f, 0xd56, 0xa5a, 0xb53, 0x859, 0x950,
		0x7c0, 0x6c9, 0x5c3, 0x4ca, 0x3c6, 0x2cf, 0x1c5, 0xcc ,
		0xfcc, 0xec5, 0xdcf, 0xcc6, 0xbca, 0xac3, 0x9c9, 0x8c0,
		0x8c0, 0x9c9, 0xac3, 0xbca, 0xcc6, 0xdcf, 0xec5, 0xfcc,
		0xcc , 0x1c5, 0x2cf, 0x3c6, 0x4ca, 0x5c3, 0x6c9, 0x7c0,
		0x950, 0x859, 0xb53, 0xa5a, 0xd56, 0xc5f, 0xf55, 0xe5c,
		0x15c, 0x55 , 0x35f, 0x256, 0x55a, 0x453, 0x759, 0x650,
		0xaf0, 0xbf9, 0x8f3, 0x9fa, 0xef6, 0xfff, 0xcf5, 0xdfc,
		0x2fc, 0x3f5, 0xff , 0x1f6, 0x6fa, 0x7f3, 0x4f9, 0x5f0,
		0xb60, 0xa69, 0x963, 0x86a, 0xf66, 0xe6f, 0xd65, 0xc6c,
		0x36c, 0x265, 0x16f, 0x66 , 0x76a, 0x663, 0x569, 0x460,
		0xca0, 0xda9, 0xea3, 0xfaa, 0x8a6, 0x9af, 0xaa5, 0xbac,
		0x4ac, 0x5a5, 0x6af, 0x7a6, 0xaa , 0x1a3, 0x2a9, 0x3a0,
		0xd30, 0xc39, 0xf33, 0xe3a, 0x936, 0x83f, 0xb35, 0xa3c,
		0x53c, 0x435, 0x73f, 0x636, 0x13a, 0x33 , 0x339, 0x230,
		0xe90, 0xf99, 0xc93, 0xd9a, 0xa96, 0xb9f, 0x895, 0x99c,
		0x69c, 0x795, 0x49f, 0x596, 0x29a, 0x393, 0x99 , 0x190,
		0xf00, 0xe09, 0xd03, 0xc0a, 0xb06, 0xa0f, 0x905, 0x80c,
		0x70c, 0x605, 0x50f, 0x406, 0x30a, 0x203, 0x109, 0x0   };


		//int i,ntri = 0;

		cubeindex = 0;

		if (g[i+j*y+k*y*z].val[0] < iso) cubeindex |= 1;
		if (g[i+j*y+k*y*z].val[1] < iso) cubeindex |= 2;
		if (g[i+j*y+k*y*z].val[2] < iso) cubeindex |= 4;
		if (g[i+j*y+k*y*z].val[3] < iso) cubeindex |= 8;
		if (g[i+j*y+k*y*z].val[4] < iso) cubeindex |= 16;
		if (g[i+j*y+k*y*z].val[5] < iso) cubeindex |= 32;
		if (g[i+j*y+k*y*z].val[6] < iso) cubeindex |= 64;
		if (g[i+j*y+k*y*z].val[7] < iso) cubeindex |= 128;
		

		int triTable[256][16] =
			{{-1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{0, 8, 3, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{0, 1, 9, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{1, 8, 3, 9, 8, 1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{1, 2, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{0, 8, 3, 1, 2, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{9, 2, 10, 0, 2, 9, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{2, 8, 3, 2, 10, 8, 10, 9, 8, -1, -1, -1, -1, -1, -1, -1},
			{3, 11, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{0, 11, 2, 8, 11, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{1, 9, 0, 2, 3, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{1, 11, 2, 1, 9, 11, 9, 8, 11, -1, -1, -1, -1, -1, -1, -1},
			{3, 10, 1, 11, 10, 3, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{0, 10, 1, 0, 8, 10, 8, 11, 10, -1, -1, -1, -1, -1, -1, -1},
			{3, 9, 0, 3, 11, 9, 11, 10, 9, -1, -1, -1, -1, -1, -1, -1},
			{9, 8, 10, 10, 8, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{4, 7, 8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{4, 3, 0, 7, 3, 4, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{0, 1, 9, 8, 4, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{4, 1, 9, 4, 7, 1, 7, 3, 1, -1, -1, -1, -1, -1, -1, -1},
			{1, 2, 10, 8, 4, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{3, 4, 7, 3, 0, 4, 1, 2, 10, -1, -1, -1, -1, -1, -1, -1},
			{9, 2, 10, 9, 0, 2, 8, 4, 7, -1, -1, -1, -1, -1, -1, -1},
			{2, 10, 9, 2, 9, 7, 2, 7, 3, 7, 9, 4, -1, -1, -1, -1},
			{8, 4, 7, 3, 11, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{11, 4, 7, 11, 2, 4, 2, 0, 4, -1, -1, -1, -1, -1, -1, -1},
			{9, 0, 1, 8, 4, 7, 2, 3, 11, -1, -1, -1, -1, -1, -1, -1},
			{4, 7, 11, 9, 4, 11, 9, 11, 2, 9, 2, 1, -1, -1, -1, -1},
			{3, 10, 1, 3, 11, 10, 7, 8, 4, -1, -1, -1, -1, -1, -1, -1},
			{1, 11, 10, 1, 4, 11, 1, 0, 4, 7, 11, 4, -1, -1, -1, -1},
			{4, 7, 8, 9, 0, 11, 9, 11, 10, 11, 0, 3, -1, -1, -1, -1},
			{4, 7, 11, 4, 11, 9, 9, 11, 10, -1, -1, -1, -1, -1, -1, -1},
			{9, 5, 4, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{9, 5, 4, 0, 8, 3, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{0, 5, 4, 1, 5, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{8, 5, 4, 8, 3, 5, 3, 1, 5, -1, -1, -1, -1, -1, -1, -1},
			{1, 2, 10, 9, 5, 4, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{3, 0, 8, 1, 2, 10, 4, 9, 5, -1, -1, -1, -1, -1, -1, -1},
			{5, 2, 10, 5, 4, 2, 4, 0, 2, -1, -1, -1, -1, -1, -1, -1},
			{2, 10, 5, 3, 2, 5, 3, 5, 4, 3, 4, 8, -1, -1, -1, -1},
			{9, 5, 4, 2, 3, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{0, 11, 2, 0, 8, 11, 4, 9, 5, -1, -1, -1, -1, -1, -1, -1},
			{0, 5, 4, 0, 1, 5, 2, 3, 11, -1, -1, -1, -1, -1, -1, -1},
			{2, 1, 5, 2, 5, 8, 2, 8, 11, 4, 8, 5, -1, -1, -1, -1},
			{10, 3, 11, 10, 1, 3, 9, 5, 4, -1, -1, -1, -1, -1, -1, -1},
			{4, 9, 5, 0, 8, 1, 8, 10, 1, 8, 11, 10, -1, -1, -1, -1},
			{5, 4, 0, 5, 0, 11, 5, 11, 10, 11, 0, 3, -1, -1, -1, -1},
			{5, 4, 8, 5, 8, 10, 10, 8, 11, -1, -1, -1, -1, -1, -1, -1},
			{9, 7, 8, 5, 7, 9, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{9, 3, 0, 9, 5, 3, 5, 7, 3, -1, -1, -1, -1, -1, -1, -1},
			{0, 7, 8, 0, 1, 7, 1, 5, 7, -1, -1, -1, -1, -1, -1, -1},
			{1, 5, 3, 3, 5, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{9, 7, 8, 9, 5, 7, 10, 1, 2, -1, -1, -1, -1, -1, -1, -1},
			{10, 1, 2, 9, 5, 0, 5, 3, 0, 5, 7, 3, -1, -1, -1, -1},
			{8, 0, 2, 8, 2, 5, 8, 5, 7, 10, 5, 2, -1, -1, -1, -1},
			{2, 10, 5, 2, 5, 3, 3, 5, 7, -1, -1, -1, -1, -1, -1, -1},
			{7, 9, 5, 7, 8, 9, 3, 11, 2, -1, -1, -1, -1, -1, -1, -1},
			{9, 5, 7, 9, 7, 2, 9, 2, 0, 2, 7, 11, -1, -1, -1, -1},
			{2, 3, 11, 0, 1, 8, 1, 7, 8, 1, 5, 7, -1, -1, -1, -1},
			{11, 2, 1, 11, 1, 7, 7, 1, 5, -1, -1, -1, -1, -1, -1, -1},
			{9, 5, 8, 8, 5, 7, 10, 1, 3, 10, 3, 11, -1, -1, -1, -1},
			{5, 7, 0, 5, 0, 9, 7, 11, 0, 1, 0, 10, 11, 10, 0, -1},
			{11, 10, 0, 11, 0, 3, 10, 5, 0, 8, 0, 7, 5, 7, 0, -1},
			{11, 10, 5, 7, 11, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{10, 6, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{0, 8, 3, 5, 10, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{9, 0, 1, 5, 10, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{1, 8, 3, 1, 9, 8, 5, 10, 6, -1, -1, -1, -1, -1, -1, -1},
			{1, 6, 5, 2, 6, 1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{1, 6, 5, 1, 2, 6, 3, 0, 8, -1, -1, -1, -1, -1, -1, -1},
			{9, 6, 5, 9, 0, 6, 0, 2, 6, -1, -1, -1, -1, -1, -1, -1},
			{5, 9, 8, 5, 8, 2, 5, 2, 6, 3, 2, 8, -1, -1, -1, -1},
			{2, 3, 11, 10, 6, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{11, 0, 8, 11, 2, 0, 10, 6, 5, -1, -1, -1, -1, -1, -1, -1},
			{0, 1, 9, 2, 3, 11, 5, 10, 6, -1, -1, -1, -1, -1, -1, -1},
			{5, 10, 6, 1, 9, 2, 9, 11, 2, 9, 8, 11, -1, -1, -1, -1},
			{6, 3, 11, 6, 5, 3, 5, 1, 3, -1, -1, -1, -1, -1, -1, -1},
			{0, 8, 11, 0, 11, 5, 0, 5, 1, 5, 11, 6, -1, -1, -1, -1},
			{3, 11, 6, 0, 3, 6, 0, 6, 5, 0, 5, 9, -1, -1, -1, -1},
			{6, 5, 9, 6, 9, 11, 11, 9, 8, -1, -1, -1, -1, -1, -1, -1},
			{5, 10, 6, 4, 7, 8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{4, 3, 0, 4, 7, 3, 6, 5, 10, -1, -1, -1, -1, -1, -1, -1},
			{1, 9, 0, 5, 10, 6, 8, 4, 7, -1, -1, -1, -1, -1, -1, -1},
			{10, 6, 5, 1, 9, 7, 1, 7, 3, 7, 9, 4, -1, -1, -1, -1},
			{6, 1, 2, 6, 5, 1, 4, 7, 8, -1, -1, -1, -1, -1, -1, -1},
			{1, 2, 5, 5, 2, 6, 3, 0, 4, 3, 4, 7, -1, -1, -1, -1},
			{8, 4, 7, 9, 0, 5, 0, 6, 5, 0, 2, 6, -1, -1, -1, -1},
			{7, 3, 9, 7, 9, 4, 3, 2, 9, 5, 9, 6, 2, 6, 9, -1},
			{3, 11, 2, 7, 8, 4, 10, 6, 5, -1, -1, -1, -1, -1, -1, -1},
			{5, 10, 6, 4, 7, 2, 4, 2, 0, 2, 7, 11, -1, -1, -1, -1},
			{0, 1, 9, 4, 7, 8, 2, 3, 11, 5, 10, 6, -1, -1, -1, -1},
			{9, 2, 1, 9, 11, 2, 9, 4, 11, 7, 11, 4, 5, 10, 6, -1},
			{8, 4, 7, 3, 11, 5, 3, 5, 1, 5, 11, 6, -1, -1, -1, -1},
			{5, 1, 11, 5, 11, 6, 1, 0, 11, 7, 11, 4, 0, 4, 11, -1},
			{0, 5, 9, 0, 6, 5, 0, 3, 6, 11, 6, 3, 8, 4, 7, -1},
			{6, 5, 9, 6, 9, 11, 4, 7, 9, 7, 11, 9, -1, -1, -1, -1},
			{10, 4, 9, 6, 4, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{4, 10, 6, 4, 9, 10, 0, 8, 3, -1, -1, -1, -1, -1, -1, -1},
			{10, 0, 1, 10, 6, 0, 6, 4, 0, -1, -1, -1, -1, -1, -1, -1},
			{8, 3, 1, 8, 1, 6, 8, 6, 4, 6, 1, 10, -1, -1, -1, -1},
			{1, 4, 9, 1, 2, 4, 2, 6, 4, -1, -1, -1, -1, -1, -1, -1},
			{3, 0, 8, 1, 2, 9, 2, 4, 9, 2, 6, 4, -1, -1, -1, -1},
			{0, 2, 4, 4, 2, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{8, 3, 2, 8, 2, 4, 4, 2, 6, -1, -1, -1, -1, -1, -1, -1},
			{10, 4, 9, 10, 6, 4, 11, 2, 3, -1, -1, -1, -1, -1, -1, -1},
			{0, 8, 2, 2, 8, 11, 4, 9, 10, 4, 10, 6, -1, -1, -1, -1},
			{3, 11, 2, 0, 1, 6, 0, 6, 4, 6, 1, 10, -1, -1, -1, -1},
			{6, 4, 1, 6, 1, 10, 4, 8, 1, 2, 1, 11, 8, 11, 1, -1},
			{9, 6, 4, 9, 3, 6, 9, 1, 3, 11, 6, 3, -1, -1, -1, -1},
			{8, 11, 1, 8, 1, 0, 11, 6, 1, 9, 1, 4, 6, 4, 1, -1},
			{3, 11, 6, 3, 6, 0, 0, 6, 4, -1, -1, -1, -1, -1, -1, -1},
			{6, 4, 8, 11, 6, 8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{7, 10, 6, 7, 8, 10, 8, 9, 10, -1, -1, -1, -1, -1, -1, -1},
			{0, 7, 3, 0, 10, 7, 0, 9, 10, 6, 7, 10, -1, -1, -1, -1},
			{10, 6, 7, 1, 10, 7, 1, 7, 8, 1, 8, 0, -1, -1, -1, -1},
			{10, 6, 7, 10, 7, 1, 1, 7, 3, -1, -1, -1, -1, -1, -1, -1},
			{1, 2, 6, 1, 6, 8, 1, 8, 9, 8, 6, 7, -1, -1, -1, -1},
			{2, 6, 9, 2, 9, 1, 6, 7, 9, 0, 9, 3, 7, 3, 9, -1},
			{7, 8, 0, 7, 0, 6, 6, 0, 2, -1, -1, -1, -1, -1, -1, -1},
			{7, 3, 2, 6, 7, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{2, 3, 11, 10, 6, 8, 10, 8, 9, 8, 6, 7, -1, -1, -1, -1},
			{2, 0, 7, 2, 7, 11, 0, 9, 7, 6, 7, 10, 9, 10, 7, -1},
			{1, 8, 0, 1, 7, 8, 1, 10, 7, 6, 7, 10, 2, 3, 11, -1},
			{11, 2, 1, 11, 1, 7, 10, 6, 1, 6, 7, 1, -1, -1, -1, -1},
			{8, 9, 6, 8, 6, 7, 9, 1, 6, 11, 6, 3, 1, 3, 6, -1},
			{0, 9, 1, 11, 6, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{7, 8, 0, 7, 0, 6, 3, 11, 0, 11, 6, 0, -1, -1, -1, -1},
			{7, 11, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{7, 6, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{3, 0, 8, 11, 7, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{0, 1, 9, 11, 7, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{8, 1, 9, 8, 3, 1, 11, 7, 6, -1, -1, -1, -1, -1, -1, -1},
			{10, 1, 2, 6, 11, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{1, 2, 10, 3, 0, 8, 6, 11, 7, -1, -1, -1, -1, -1, -1, -1},
			{2, 9, 0, 2, 10, 9, 6, 11, 7, -1, -1, -1, -1, -1, -1, -1},
			{6, 11, 7, 2, 10, 3, 10, 8, 3, 10, 9, 8, -1, -1, -1, -1},
			{7, 2, 3, 6, 2, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{7, 0, 8, 7, 6, 0, 6, 2, 0, -1, -1, -1, -1, -1, -1, -1},
			{2, 7, 6, 2, 3, 7, 0, 1, 9, -1, -1, -1, -1, -1, -1, -1},
			{1, 6, 2, 1, 8, 6, 1, 9, 8, 8, 7, 6, -1, -1, -1, -1},
			{10, 7, 6, 10, 1, 7, 1, 3, 7, -1, -1, -1, -1, -1, -1, -1},
			{10, 7, 6, 1, 7, 10, 1, 8, 7, 1, 0, 8, -1, -1, -1, -1},
			{0, 3, 7, 0, 7, 10, 0, 10, 9, 6, 10, 7, -1, -1, -1, -1},
			{7, 6, 10, 7, 10, 8, 8, 10, 9, -1, -1, -1, -1, -1, -1, -1},
			{6, 8, 4, 11, 8, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{3, 6, 11, 3, 0, 6, 0, 4, 6, -1, -1, -1, -1, -1, -1, -1},
			{8, 6, 11, 8, 4, 6, 9, 0, 1, -1, -1, -1, -1, -1, -1, -1},
			{9, 4, 6, 9, 6, 3, 9, 3, 1, 11, 3, 6, -1, -1, -1, -1},
			{6, 8, 4, 6, 11, 8, 2, 10, 1, -1, -1, -1, -1, -1, -1, -1},
			{1, 2, 10, 3, 0, 11, 0, 6, 11, 0, 4, 6, -1, -1, -1, -1},
			{4, 11, 8, 4, 6, 11, 0, 2, 9, 2, 10, 9, -1, -1, -1, -1},
			{10, 9, 3, 10, 3, 2, 9, 4, 3, 11, 3, 6, 4, 6, 3, -1},
			{8, 2, 3, 8, 4, 2, 4, 6, 2, -1, -1, -1, -1, -1, -1, -1},
			{0, 4, 2, 4, 6, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{1, 9, 0, 2, 3, 4, 2, 4, 6, 4, 3, 8, -1, -1, -1, -1},
			{1, 9, 4, 1, 4, 2, 2, 4, 6, -1, -1, -1, -1, -1, -1, -1},
			{8, 1, 3, 8, 6, 1, 8, 4, 6, 6, 10, 1, -1, -1, -1, -1},
			{10, 1, 0, 10, 0, 6, 6, 0, 4, -1, -1, -1, -1, -1, -1, -1},
			{4, 6, 3, 4, 3, 8, 6, 10, 3, 0, 3, 9, 10, 9, 3, -1},
			{10, 9, 4, 6, 10, 4, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{4, 9, 5, 7, 6, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{0, 8, 3, 4, 9, 5, 11, 7, 6, -1, -1, -1, -1, -1, -1, -1},
			{5, 0, 1, 5, 4, 0, 7, 6, 11, -1, -1, -1, -1, -1, -1, -1},
			{11, 7, 6, 8, 3, 4, 3, 5, 4, 3, 1, 5, -1, -1, -1, -1},
			{9, 5, 4, 10, 1, 2, 7, 6, 11, -1, -1, -1, -1, -1, -1, -1},
			{6, 11, 7, 1, 2, 10, 0, 8, 3, 4, 9, 5, -1, -1, -1, -1},
			{7, 6, 11, 5, 4, 10, 4, 2, 10, 4, 0, 2, -1, -1, -1, -1},
			{3, 4, 8, 3, 5, 4, 3, 2, 5, 10, 5, 2, 11, 7, 6, -1},
			{7, 2, 3, 7, 6, 2, 5, 4, 9, -1, -1, -1, -1, -1, -1, -1},
			{9, 5, 4, 0, 8, 6, 0, 6, 2, 6, 8, 7, -1, -1, -1, -1},
			{3, 6, 2, 3, 7, 6, 1, 5, 0, 5, 4, 0, -1, -1, -1, -1},
			{6, 2, 8, 6, 8, 7, 2, 1, 8, 4, 8, 5, 1, 5, 8, -1},
			{9, 5, 4, 10, 1, 6, 1, 7, 6, 1, 3, 7, -1, -1, -1, -1},
			{1, 6, 10, 1, 7, 6, 1, 0, 7, 8, 7, 0, 9, 5, 4, -1},
			{4, 0, 10, 4, 10, 5, 0, 3, 10, 6, 10, 7, 3, 7, 10, -1},
			{7, 6, 10, 7, 10, 8, 5, 4, 10, 4, 8, 10, -1, -1, -1, -1},
			{6, 9, 5, 6, 11, 9, 11, 8, 9, -1, -1, -1, -1, -1, -1, -1},
			{3, 6, 11, 0, 6, 3, 0, 5, 6, 0, 9, 5, -1, -1, -1, -1},
			{0, 11, 8, 0, 5, 11, 0, 1, 5, 5, 6, 11, -1, -1, -1, -1},
			{6, 11, 3, 6, 3, 5, 5, 3, 1, -1, -1, -1, -1, -1, -1, -1},
			{1, 2, 10, 9, 5, 11, 9, 11, 8, 11, 5, 6, -1, -1, -1, -1},
			{0, 11, 3, 0, 6, 11, 0, 9, 6, 5, 6, 9, 1, 2, 10, -1},
			{11, 8, 5, 11, 5, 6, 8, 0, 5, 10, 5, 2, 0, 2, 5, -1},
			{6, 11, 3, 6, 3, 5, 2, 10, 3, 10, 5, 3, -1, -1, -1, -1},
			{5, 8, 9, 5, 2, 8, 5, 6, 2, 3, 8, 2, -1, -1, -1, -1},
			{9, 5, 6, 9, 6, 0, 0, 6, 2, -1, -1, -1, -1, -1, -1, -1},
			{1, 5, 8, 1, 8, 0, 5, 6, 8, 3, 8, 2, 6, 2, 8, -1},
			{1, 5, 6, 2, 1, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{1, 3, 6, 1, 6, 10, 3, 8, 6, 5, 6, 9, 8, 9, 6, -1},
			{10, 1, 0, 10, 0, 6, 9, 5, 0, 5, 6, 0, -1, -1, -1, -1},
			{0, 3, 8, 5, 6, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{10, 5, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{11, 5, 10, 7, 5, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{11, 5, 10, 11, 7, 5, 8, 3, 0, -1, -1, -1, -1, -1, -1, -1},
			{5, 11, 7, 5, 10, 11, 1, 9, 0, -1, -1, -1, -1, -1, -1, -1},
			{10, 7, 5, 10, 11, 7, 9, 8, 1, 8, 3, 1, -1, -1, -1, -1},
			{11, 1, 2, 11, 7, 1, 7, 5, 1, -1, -1, -1, -1, -1, -1, -1},
			{0, 8, 3, 1, 2, 7, 1, 7, 5, 7, 2, 11, -1, -1, -1, -1},
			{9, 7, 5, 9, 2, 7, 9, 0, 2, 2, 11, 7, -1, -1, -1, -1},
			{7, 5, 2, 7, 2, 11, 5, 9, 2, 3, 2, 8, 9, 8, 2, -1},
			{2, 5, 10, 2, 3, 5, 3, 7, 5, -1, -1, -1, -1, -1, -1, -1},
			{8, 2, 0, 8, 5, 2, 8, 7, 5, 10, 2, 5, -1, -1, -1, -1},
			{9, 0, 1, 5, 10, 3, 5, 3, 7, 3, 10, 2, -1, -1, -1, -1},
			{9, 8, 2, 9, 2, 1, 8, 7, 2, 10, 2, 5, 7, 5, 2, -1},
			{1, 3, 5, 3, 7, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{0, 8, 7, 0, 7, 1, 1, 7, 5, -1, -1, -1, -1, -1, -1, -1},
			{9, 0, 3, 9, 3, 5, 5, 3, 7, -1, -1, -1, -1, -1, -1, -1},
			{9, 8, 7, 5, 9, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{5, 8, 4, 5, 10, 8, 10, 11, 8, -1, -1, -1, -1, -1, -1, -1},
			{5, 0, 4, 5, 11, 0, 5, 10, 11, 11, 3, 0, -1, -1, -1, -1},
			{0, 1, 9, 8, 4, 10, 8, 10, 11, 10, 4, 5, -1, -1, -1, -1},
			{10, 11, 4, 10, 4, 5, 11, 3, 4, 9, 4, 1, 3, 1, 4, -1},
			{2, 5, 1, 2, 8, 5, 2, 11, 8, 4, 5, 8, -1, -1, -1, -1},
			{0, 4, 11, 0, 11, 3, 4, 5, 11, 2, 11, 1, 5, 1, 11, -1},
			{0, 2, 5, 0, 5, 9, 2, 11, 5, 4, 5, 8, 11, 8, 5, -1},
			{9, 4, 5, 2, 11, 3, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{2, 5, 10, 3, 5, 2, 3, 4, 5, 3, 8, 4, -1, -1, -1, -1},
			{5, 10, 2, 5, 2, 4, 4, 2, 0, -1, -1, -1, -1, -1, -1, -1},
			{3, 10, 2, 3, 5, 10, 3, 8, 5, 4, 5, 8, 0, 1, 9, -1},
			{5, 10, 2, 5, 2, 4, 1, 9, 2, 9, 4, 2, -1, -1, -1, -1},
			{8, 4, 5, 8, 5, 3, 3, 5, 1, -1, -1, -1, -1, -1, -1, -1},
			{0, 4, 5, 1, 0, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{8, 4, 5, 8, 5, 3, 9, 0, 5, 0, 3, 5, -1, -1, -1, -1},
			{9, 4, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{4, 11, 7, 4, 9, 11, 9, 10, 11, -1, -1, -1, -1, -1, -1, -1},
			{0, 8, 3, 4, 9, 7, 9, 11, 7, 9, 10, 11, -1, -1, -1, -1},
			{1, 10, 11, 1, 11, 4, 1, 4, 0, 7, 4, 11, -1, -1, -1, -1},
			{3, 1, 4, 3, 4, 8, 1, 10, 4, 7, 4, 11, 10, 11, 4, -1},
			{4, 11, 7, 9, 11, 4, 9, 2, 11, 9, 1, 2, -1, -1, -1, -1},
			{9, 7, 4, 9, 11, 7, 9, 1, 11, 2, 11, 1, 0, 8, 3, -1},
			{11, 7, 4, 11, 4, 2, 2, 4, 0, -1, -1, -1, -1, -1, -1, -1},
			{11, 7, 4, 11, 4, 2, 8, 3, 4, 3, 2, 4, -1, -1, -1, -1},
			{2, 9, 10, 2, 7, 9, 2, 3, 7, 7, 4, 9, -1, -1, -1, -1},
			{9, 10, 7, 9, 7, 4, 10, 2, 7, 8, 7, 0, 2, 0, 7, -1},
			{3, 7, 10, 3, 10, 2, 7, 4, 10, 1, 10, 0, 4, 0, 10, -1},
			{1, 10, 2, 8, 7, 4, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{4, 9, 1, 4, 1, 7, 7, 1, 3, -1, -1, -1, -1, -1, -1, -1},
			{4, 9, 1, 4, 1, 7, 0, 8, 1, 8, 7, 1, -1, -1, -1, -1},
			{4, 0, 3, 7, 4, 3, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{4, 8, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{9, 10, 8, 10, 11, 8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{3, 0, 9, 3, 9, 11, 11, 9, 10, -1, -1, -1, -1, -1, -1, -1},
			{0, 1, 10, 0, 10, 8, 8, 10, 11, -1, -1, -1, -1, -1, -1, -1},
			{3, 1, 10, 11, 3, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{1, 2, 11, 1, 11, 9, 9, 11, 8, -1, -1, -1, -1, -1, -1, -1},
			{3, 0, 9, 3, 9, 11, 1, 2, 9, 2, 11, 9, -1, -1, -1, -1},
			{0, 2, 11, 8, 0, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{3, 2, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{2, 3, 8, 2, 8, 10, 10, 8, 9, -1, -1, -1, -1, -1, -1, -1},
			{9, 10, 2, 0, 9, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{2, 3, 8, 2, 8, 10, 0, 1, 8, 1, 10, 8, -1, -1, -1, -1},
			{1, 10, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{1, 3, 8, 9, 1, 8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{0, 9, 1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{0, 3, 8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{-1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1}};

	   if (edgeTable[cubeindex] == 0)
	      return;
	   /* Find the vertices where the surface intersects the cube */
	   if (edgeTable[cubeindex] & 1) {
	      vertlist[i+j*y+k*y*z+0] = VertexInterp(iso,g[i+j*y+k*y*z].p[0],g[i+j*y+k*y*z].p[1],g[i+j*y+k*y*z].val[0],g[i+j*y+k*y*z].val[1]);
	   }
	   if (edgeTable[cubeindex] & 2) {
	      vertlist[i+j*y+k*y*z+1] = VertexInterp(iso,g[i+j*y+k*y*z].p[1],g[i+j*y+k*y*z].p[2],g[i+j*y+k*y*z].val[1],g[i+j*y+k*y*z].val[2]);
	   }
	   if (edgeTable[cubeindex] & 4) {
	      vertlist[i+j*y+k*y*z+2] = VertexInterp(iso,g[i+j*y+k*y*z].p[2],g[i+j*y+k*y*z].p[3],g[i+j*y+k*y*z].val[2],g[i+j*y+k*y*z].val[3]);
	   }
	   if (edgeTable[cubeindex] & 8) {
	      vertlist[i+j*y+k*y*z+3] = VertexInterp(iso,g[i+j*y+k*y*z].p[3],g[i+j*y+k*y*z].p[0],g[i+j*y+k*y*z].val[3],g[i+j*y+k*y*z].val[0]);
	   }
	   if (edgeTable[cubeindex] & 16) {
	      vertlist[i+j*y+k*y*z+4] = VertexInterp(iso,g[i+j*y+k*y*z].p[4],g[i+j*y+k*y*z].p[5],g[i+j*y+k*y*z].val[4],g[i+j*y+k*y*z].val[5]);
	   }
	   if (edgeTable[cubeindex] & 32) {
	      vertlist[i+j*y+k*y*z+5] = VertexInterp(iso,g[i+j*y+k*y*z].p[5],g[i+j*y+k*y*z].p[6],g[i+j*y+k*y*z].val[5],g[i+j*y+k*y*z].val[6]);
	   }
	   if (edgeTable[cubeindex] & 64) {
	      vertlist[i+j*y+k*y*z+6] = VertexInterp(iso,g[i+j*y+k*y*z].p[6],g[i+j*y+k*y*z].p[7],g[i+j*y+k*y*z].val[6],g[i+j*y+k*y*z].val[7]);
	   }
	   if (edgeTable[cubeindex] & 128) {
	      vertlist[i+j*y+k*y*z+7] = VertexInterp(iso,g[i+j*y+k*y*z].p[7],g[i+j*y+k*y*z].p[4],g[i+j*y+k*y*z].val[7],g[i+j*y+k*y*z].val[4]);
	   }
	   if (edgeTable[cubeindex] & 256) {
	      vertlist[i+j*y+k*y*z+8] = VertexInterp(iso,g[i+j*y+k*y*z].p[0],g[i+j*y+k*y*z].p[4],g[i+j*y+k*y*z].val[0],g[i+j*y+k*y*z].val[4]);
	   }
	   if (edgeTable[cubeindex] & 512) {
	      vertlist[i+j*y+k*y*z+9] = VertexInterp(iso,g[i+j*y+k*y*z].p[1],g[i+j*y+k*y*z].p[5],g[i+j*y+k*y*z].val[1],g[i+j*y+k*y*z].val[5]);
	   }
	   if (edgeTable[cubeindex] & 1024) {
	      vertlist[i+j*y+k*y*z+10] = VertexInterp(iso,g[i+j*y+k*y*z].p[2],g[i+j*y+k*y*z].p[6],g[i+j*y+k*y*z].val[2],g[i+j*y+k*y*z].val[6]);
	   }
	   if (edgeTable[cubeindex] & 2048) {
	      vertlist[i+j*y+k*y*z+11] = VertexInterp(iso,g[i+j*y+k*y*z].p[3],g[i+j*y+k*y*z].p[7],g[i+j*y+k*y*z].val[3],g[i+j*y+k*y*z].val[7]);
	   }
	  // printf("hasta aqui llenaga \n");
	   //separo memoria para el host y para los triagulos, despues 
	   //creo una memorai temporal para el vertlist y almaceno en 
	   //un device memorry los triangulos.
	}
}


void printGrid(string a, GRIDCELL * g, int tam)
{
	cout<<a;
	for(int i =0; i<tam ;i++)
		for(int j=0;j<8;j++)
			//printf("%f  %f  %f \n", g[i].p[j].x ,g[i].p[j].y,g[i].p[j].z);
		      printf("%f \n", g[i].val[j]);		
}

void printTriangles(string a , TRIANGLE * t, int tam)
{
	cout<<a;
	for(int i =0; i<tam ;i++)
	{
		for (int k=0;k<3;k++)  
			cout<<t[i].p[k].x<<" "<<t[i].p[k].y<<" "<<t[i].p[k].z<<endl;
	}
}

int main(int argc, char *argv[])
{
	int i,j,k,c;
	int numtri=0;
	int ***data;
	FILE *fptr;
	int N= (NX*NY*NZ);
	cout<<N<<endl; //return 1;
	int THREADS_PER_BLOCK =8;
	int themin=255;
	int themax=0;
	int isolevel=80;
	//const char* FILENAME = "mri.raw";
	//assingMem(data);
	//readFile(fptr,FILENAME,themin, themax,data);
	// Malloc the volumetric data, hardwired size!
	data = (int***)malloc(NX*sizeof(int **));
	for (i=0;i<NX;i++)
		data[i] = (int**)malloc(NY*sizeof(int *));
	for (i=0;i<NX;i++)
		for (j=0;j<NY;j++)
			data[i][j] = (int*)malloc(NZ*sizeof(int));

	//cout<<data[199][60][0]<<endl;
	// Open and read the raw data
	fprintf(stderr,"Reading data ...\n");
	if ((fptr = fopen(argv[argc-1],"rb")) == NULL) {
		fprintf(stderr,"File open failed\n");
		exit(-1);
	}
	cout<<"llega"<<endl;
	for (k=0;k<NZ;k++) {
		for (j=0;j<NY;j++) {
			for (i=0;i<NX;i++) {
				if ((c = fgetc(fptr)) == EOF) {
					fprintf(stderr,"Unexpected end of file\n");
					exit(-1);
				}
				data[i][j][k] = c;
				//cout<<i<<" "<<j <<" "<<k <<" data : "<<data[i][j][k]<<endl;
				if (c > themax)
					themax = c;
				if (c < themin)
					themin = c;
			}
		}
	}
	fclose(fptr);
	fprintf(stderr,"Volumetric data range: %d -> %d\n",themin,themax);

	int sizeGRID= N*sizeof(GRIDCELL);
	int sizeTRI = N*sizeof(TRIANGLE); 

	cout<<"sizeGRID "<<sizeGRID<<endl;
	cout<<"sizeTRI "<<sizeTRI<<endl;

	GRIDCELL * vectGrids;
	GRIDCELL * d_vectGrids;
	TRIANGLE * vectTriangles;
	TRIANGLE * d_vectTriangles;

	vectGrids = (GRIDCELL *)malloc(sizeGRID);
	vectTriangles= (TRIANGLE *)malloc(sizeTRI); 
	constructCubes(vectGrids,data,N);
	/*
		typedef struct {
		XYZ p[3];     
		//  XYZ c;    
		//  XYZ n[3]; 
		} TRIANGLE;

		typedef struct {
		XYZ p[8];
		double val[8];
		} GRIDCELL;	
	*/	
	size_t available, total;
	hipMemGetInfo(&available, &total);
		cout<<"available:  " << available<<" total:  "<<total <<endl;
	
	hipMalloc((void **)&d_vectGrids, sizeGRID);
	hipMemcpy(d_vectGrids,vectGrids, sizeGRID, hipMemcpyHostToDevice);	
	cout<<"asignacion en RAM"<<endl;
 
 	hipMalloc((void **)&d_vectTriangles, sizeTRI);
	hipMemcpy(d_vectTriangles,vectTriangles,sizeTRI,hipMemcpyHostToDevice);
	cout<<"mem of grid "<<d_vectGrids<<endl;
	cout<<"mem of triangulos "<<d_vectTriangles<<endl;
	
	cout<<"separa memoria sin problemas"<<endl;
	//printGrid("imprimiendo Grid inicial en Host \n ",vectGrids,N);
	

	hipEvent_t start, stop;
	float elapsedTime;
	hipEventCreate(&start);


	int x = NX; int y = NY ; int z = NZ;
	int blockX= (NX + THREADS_PER_BLOCK -1)/THREADS_PER_BLOCK;
	int blockY= (NY + THREADS_PER_BLOCK -1)/THREADS_PER_BLOCK;
	int blockZ= (NZ + THREADS_PER_BLOCK -1)/THREADS_PER_BLOCK;
	cout<<"blocks : "<<blockX<<" threds:  "<<THREADS_PER_BLOCK<<endl;
	cout<<"blocks : "<<blockY<<" threds:  "<<THREADS_PER_BLOCK<<endl;
	cout<<"blocks : "<<blockZ<<" threds:  "<<THREADS_PER_BLOCK<<endl;
	//int blocks= (10 + THREADS_PER_BLOCK -1)/THREADS_PER_BLOCK;
	/*cout<<"blocks : \n"<<blocks<<"\n threds: \n "<<THREADS_PER_BLOCK<<endl; */
	dim3 dimGrid(blockX, blockY, blockZ);
	dim3 dimBlock(THREADS_PER_BLOCK,THREADS_PER_BLOCK, THREADS_PER_BLOCK);
	hipEventRecord(start,0);
	isolevel=10;
		//copyGRID1<<<dimGrid,dimBlock>>>(d_res,d_vectGrid,x,y,z);
		PolygoniseCube<<<dimGrid,dimBlock>>>(d_vectTriangles,d_vectGrids,isolevel,x,y,z);
		//PolygoniseCube<<<blocks,THREADS_PER_BLOCK>>>(d_points,d_vectGrids,isolevel);
		//matrixAdition<<<blocks,THREADS_PER_BLOCK>>>(d_a, d_points,10);
		//matrixAditionCol<<<blocks2,THREADS_PER_BLOCK>>>( d_c, d_a, d_b,N);
	hipEventCreate(&stop);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start,stop);
	printf("Elapsed time : %f ms\n" ,elapsedTime);

	TRIANGLE * cpy_vectTriangles;
	cpy_vectTriangles= (TRIANGLE *)malloc(sizeTRI); 
	hipMemcpy(cpy_vectTriangles,d_vectTriangles, sizeTRI, hipMemcpyDeviceToHost);

	printTriangles("Printing Tringles \n",cpy_vectTriangles,numtri);

	free(vectTriangles); free(vectGrids); free(cpy_vectTriangles);
	hipFree(d_vectTriangles); hipFree(d_vectGrids);
	return 0;
}

















/*
	for(int i=0;i<N;i++)
	{
		cudaMalloc((void**)&d_p,8*sizeof(XYZ));
		//cudaMemGetInfo(&available, &total);
		//cout<<"available:  " << available<<" total:  "<<total <<endl;
		cudaMalloc((void**)&d_val,8*sizeof(double));
		//cudaMemGetInfo(&available, &total);
		//cout<<"available:  " << available<<" total:  "<<total <<endl;
		cudaMemcpy(d_p,vectGrids[i].p,8*sizeof(XYZ),cudaMemcpyHostToDevice);
		//for(int w=0;w<8;w++)
		//{
			cout<<vectGrids[i].p[w].y<<endl;
		//}
		//cudaMemGetInfo(&available, &total);
		//cout<<"available:  " << available<<" total:  "<<total <<endl;
		cudaMemcpy(d_val,vectGrids[i].val,8*sizeof(double),cudaMemcpyHostToDevice);
		//cudaMemGetInfo(&available, &total);
		//cout<<"available:  " << available<<" total:  "<<total <<endl
		cudaMemGetInfo(&available, &total);
		//cout<<"available:  " << available<<" total:  "<<total <<endl;
		cudaMemcpy(d_vectGrid[i].val, d_val, 8*sizeof(double),cudaMemcpyHostToDevice);
		cudaMemcpy(d_vectGrid[i].p, d_p, 8*sizeof(XYZ),cudaMemcpyHostToDevice);
 	 }*/
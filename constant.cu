
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string>
#include <fstream>
#include <cmath>
#include <string>
#include <iostream>

using namespace std;

typedef struct {
   float x,y,z;
} XYZ;

typedef struct {
   XYZ p[8];
   char val[8];
} GRIDCELL;

typedef struct {
   XYZ p[3];         /* Vertices */
} TRIANGLE;


#define ABS(x) (x < 0 ? -(x) : (x))

/*
#define NX 68
#define NY 256
#define NZ 256
*/

//#define NX 200//200
//#define NY 160//160
//#define NZ 160//160

//68 x 256 x 256

void fillMatrix(XYZ* a, int n)
{
   int i;
   for (i = 0; i < n; ++i)
   {
        a[i].x = 3;
        a[i].y = 2;
        a[i].z = 5;//rand()%5;
   }
}
__global__
void matrixAdition(XYZ * b, XYZ *a,int n)
{
   	int ij = threadIdx.x + blockDim.x * blockIdx.x;

	if(ij<n)
	{
		b[ij].x = a[ij].x+2;
		b[ij].y = a[ij].y+3;
		b[ij].z = a[ij].z+0;
		//printf("da %d \n" , b[ij].x);
	}
}
void printMatrix(string s, XYZ *a , int tam){
	cout<<s;
	for(int i=0;i<tam;i++)
	{
		if(a[i].x!=0 && a[i].y!=0 && a[i].z!=0)
		{
			cout<<a[i].x<<" "<<a[i].y<<" "<<a[i].z<<" ";
			cout<<endl;
		}
	}
}
void assingMem(int *** data, int NX, int NY, int NZ)
{
	int i,j;
	data = (int ***)malloc(NX*sizeof(short int **));
	for (i=0;i<NX;i++)
		data[i] = (int **)malloc(NY*sizeof(short int *));
		for (i=0;i<NX;i++)
			for (j=0;j<NY;j++)
				data[i][j] = (int *)malloc(NZ*sizeof(short int));
}
void readFile(FILE *fptr, const char * namefile , int themin , int themax, int *** data , int NX, int NY, int NZ)
{
	int i,j,k,c;
	fprintf(stderr,"Load data ...\n");
	if ((fptr = fopen(namefile,"rb")) == NULL) {
		fprintf(stderr,"Error al leer archivo\n");
		exit(-1);
	}
	for (k=0;k<NZ;k++) {
		for (j=0;j<NY;j++) {
			for (i=0;i<NX;i++) {
				if ((c = fgetc(fptr)) == EOF) {
					fprintf(stderr,"Error en tamaño\n");
					exit(-1);
				}
				data[i][j][k] = c;
				cout<<"leyendo :"<<c<<endl;
				if (c > themax)
					themax = c;
				if (c < themin)
					themin = c;
			}
		}
	}
	fclose(fptr);
	fprintf(stderr,"Rango del volumen: %d -> %d\n",themin,themax);
}

int constructCubes(GRIDCELL * vectGrids, int *** data,int NX, int NY, int NZ)
{
		int i,j,k;
		//fprintf(stderr,"Construyendo Cubos ...\n");
		int cont=0;
		for (i=0;i<NX-1;i++) {
			//cout<<i<<endl;
			//if (i % (NX/10) == 0)
				//fprintf(stderr,"   Slice %d de %d\n",i,NX);
			for (j=0;j<NY-1;j++) {
				for (k=0;k<NZ-1;k++) {
					GRIDCELL grid;
					grid.p[0].x = i;
					grid.p[0].y = j;
		         	grid.p[0].z = k;
						grid.val[0] = data[i][j][k];
		            grid.p[1].x = i+1;
		            grid.p[1].y = j;
		            grid.p[1].z = k;
						grid.val[1] = data[i+1][j][k]; /// 
		            grid.p[2].x = i+1;
		            grid.p[2].y = j+1;
		            grid.p[2].z = k;
						grid.val[2] = data[i+1][j+1][k];
		            grid.p[3].x = i;
		            grid.p[3].y = j+1;
		            grid.p[3].z = k;
						grid.val[3] = data[i][j+1][k];
		            grid.p[4].x = i;
		            grid.p[4].y = j;
		            grid.p[4].z = k+1;
						grid.val[4] = data[i][j][k+1];
		            grid.p[5].x = i+1;
		            grid.p[5].y = j;
		            grid.p[5].z = k+1;
						grid.val[5] = data[i+1][j][k+1];
		            grid.p[6].x = i+1;
		            grid.p[6].y = j+1;
		            grid.p[6].z = k+1;
						grid.val[6] = data[i+1][j+1][k+1];
		            grid.p[7].x = i;
		            grid.p[7].y = j+1;
		            grid.p[7].z = k+1;
						grid.val[7] = data[i][j+1][k+1];
					vectGrids[cont]=grid;
					cont++;
				}
			}
		}
		cout<<"numero de datos ingresados "<<cont<<endl;
		return cont;
}
__device__
XYZ VertexInterp(float isolevel,XYZ p1,XYZ p2,float valp1,float valp2)
{
   float mu;
   XYZ p;
   if (ABS(isolevel-valp1) < 0.00001)
      return(p1);
   if (ABS(isolevel-valp2) < 0.00001)
      return(p2);
   if (ABS(valp1-valp2) < 0.00001)
      return(p1);
   mu = (isolevel - valp1) / (valp2 - valp1);
   p.x = p1.x + mu * (p2.x - p1.x);
   p.y = p1.y + mu * (p2.y - p1.y);
   p.z = p1.z + mu * (p2.z - p1.z);
   return p;
}

__device__
void copyXYZ(XYZ &a, XYZ &b)
{
	a.x=b.x ; a.y=b.y ; a.z = b.z;
}

__device__
XYZ defect()
{
	XYZ a; 
	a.x=300 ; a.y=300 ; a.z = 300;
	return a;
}

__constant__ int edgeTable[256]={
		0x0  , 0x109, 0x203, 0x30a, 0x406, 0x50f, 0x605, 0x70c,
		0x80c, 0x905, 0xa0f, 0xb06, 0xc0a, 0xd03, 0xe09, 0xf00,
		0x190, 0x99 , 0x393, 0x29a, 0x596, 0x49f, 0x795, 0x69c,
		0x99c, 0x895, 0xb9f, 0xa96, 0xd9a, 0xc93, 0xf99, 0xe90,
		0x230, 0x339, 0x33 , 0x13a, 0x636, 0x73f, 0x435, 0x53c,
		0xa3c, 0xb35, 0x83f, 0x936, 0xe3a, 0xf33, 0xc39, 0xd30,
		0x3a0, 0x2a9, 0x1a3, 0xaa , 0x7a6, 0x6af, 0x5a5, 0x4ac,
		0xbac, 0xaa5, 0x9af, 0x8a6, 0xfaa, 0xea3, 0xda9, 0xca0,
		0x460, 0x569, 0x663, 0x76a, 0x66 , 0x16f, 0x265, 0x36c,
		0xc6c, 0xd65, 0xe6f, 0xf66, 0x86a, 0x963, 0xa69, 0xb60,
		0x5f0, 0x4f9, 0x7f3, 0x6fa, 0x1f6, 0xff , 0x3f5, 0x2fc,
		0xdfc, 0xcf5, 0xfff, 0xef6, 0x9fa, 0x8f3, 0xbf9, 0xaf0,
		0x650, 0x759, 0x453, 0x55a, 0x256, 0x35f, 0x55 , 0x15c,
		0xe5c, 0xf55, 0xc5f, 0xd56, 0xa5a, 0xb53, 0x859, 0x950,
		0x7c0, 0x6c9, 0x5c3, 0x4ca, 0x3c6, 0x2cf, 0x1c5, 0xcc ,
		0xfcc, 0xec5, 0xdcf, 0xcc6, 0xbca, 0xac3, 0x9c9, 0x8c0,
		0x8c0, 0x9c9, 0xac3, 0xbca, 0xcc6, 0xdcf, 0xec5, 0xfcc,
		0xcc , 0x1c5, 0x2cf, 0x3c6, 0x4ca, 0x5c3, 0x6c9, 0x7c0,
		0x950, 0x859, 0xb53, 0xa5a, 0xd56, 0xc5f, 0xf55, 0xe5c,
		0x15c, 0x55 , 0x35f, 0x256, 0x55a, 0x453, 0x759, 0x650,
		0xaf0, 0xbf9, 0x8f3, 0x9fa, 0xef6, 0xfff, 0xcf5, 0xdfc,
		0x2fc, 0x3f5, 0xff , 0x1f6, 0x6fa, 0x7f3, 0x4f9, 0x5f0,
		0xb60, 0xa69, 0x963, 0x86a, 0xf66, 0xe6f, 0xd65, 0xc6c,
		0x36c, 0x265, 0x16f, 0x66 , 0x76a, 0x663, 0x569, 0x460,
		0xca0, 0xda9, 0xea3, 0xfaa, 0x8a6, 0x9af, 0xaa5, 0xbac,
		0x4ac, 0x5a5, 0x6af, 0x7a6, 0xaa , 0x1a3, 0x2a9, 0x3a0,
		0xd30, 0xc39, 0xf33, 0xe3a, 0x936, 0x83f, 0xb35, 0xa3c,
		0x53c, 0x435, 0x73f, 0x636, 0x13a, 0x33 , 0x339, 0x230,
		0xe90, 0xf99, 0xc93, 0xd9a, 0xa96, 0xb9f, 0x895, 0x99c,
		0x69c, 0x795, 0x49f, 0x596, 0x29a, 0x393, 0x99 , 0x190,
		0xf00, 0xe09, 0xd03, 0xc0a, 0xb06, 0xa0f, 0x905, 0x80c,
		0x70c, 0x605, 0x50f, 0x406, 0x30a, 0x203, 0x109, 0x109};
__constant__ int triTable[256][16] =
			{{-1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{0, 8, 3, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{0, 1, 9, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{1, 8, 3, 9, 8, 1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{1, 2, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{0, 8, 3, 1, 2, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{9, 2, 10, 0, 2, 9, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{2, 8, 3, 2, 10, 8, 10, 9, 8, -1, -1, -1, -1, -1, -1, -1},
			{3, 11, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{0, 11, 2, 8, 11, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{1, 9, 0, 2, 3, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{1, 11, 2, 1, 9, 11, 9, 8, 11, -1, -1, -1, -1, -1, -1, -1},
			{3, 10, 1, 11, 10, 3, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{0, 10, 1, 0, 8, 10, 8, 11, 10, -1, -1, -1, -1, -1, -1, -1},
			{3, 9, 0, 3, 11, 9, 11, 10, 9, -1, -1, -1, -1, -1, -1, -1},
			{9, 8, 10, 10, 8, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{4, 7, 8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{4, 3, 0, 7, 3, 4, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{0, 1, 9, 8, 4, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{4, 1, 9, 4, 7, 1, 7, 3, 1, -1, -1, -1, -1, -1, -1, -1},
			{1, 2, 10, 8, 4, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{3, 4, 7, 3, 0, 4, 1, 2, 10, -1, -1, -1, -1, -1, -1, -1},
			{9, 2, 10, 9, 0, 2, 8, 4, 7, -1, -1, -1, -1, -1, -1, -1},
			{2, 10, 9, 2, 9, 7, 2, 7, 3, 7, 9, 4, -1, -1, -1, -1},
			{8, 4, 7, 3, 11, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{11, 4, 7, 11, 2, 4, 2, 0, 4, -1, -1, -1, -1, -1, -1, -1},
			{9, 0, 1, 8, 4, 7, 2, 3, 11, -1, -1, -1, -1, -1, -1, -1},
			{4, 7, 11, 9, 4, 11, 9, 11, 2, 9, 2, 1, -1, -1, -1, -1},
			{3, 10, 1, 3, 11, 10, 7, 8, 4, -1, -1, -1, -1, -1, -1, -1},
			{1, 11, 10, 1, 4, 11, 1, 0, 4, 7, 11, 4, -1, -1, -1, -1},
			{4, 7, 8, 9, 0, 11, 9, 11, 10, 11, 0, 3, -1, -1, -1, -1},
			{4, 7, 11, 4, 11, 9, 9, 11, 10, -1, -1, -1, -1, -1, -1, -1},
			{9, 5, 4, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{9, 5, 4, 0, 8, 3, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{0, 5, 4, 1, 5, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{8, 5, 4, 8, 3, 5, 3, 1, 5, -1, -1, -1, -1, -1, -1, -1},
			{1, 2, 10, 9, 5, 4, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{3, 0, 8, 1, 2, 10, 4, 9, 5, -1, -1, -1, -1, -1, -1, -1},
			{5, 2, 10, 5, 4, 2, 4, 0, 2, -1, -1, -1, -1, -1, -1, -1},
			{2, 10, 5, 3, 2, 5, 3, 5, 4, 3, 4, 8, -1, -1, -1, -1},
			{9, 5, 4, 2, 3, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{0, 11, 2, 0, 8, 11, 4, 9, 5, -1, -1, -1, -1, -1, -1, -1},
			{0, 5, 4, 0, 1, 5, 2, 3, 11, -1, -1, -1, -1, -1, -1, -1},
			{2, 1, 5, 2, 5, 8, 2, 8, 11, 4, 8, 5, -1, -1, -1, -1},
			{10, 3, 11, 10, 1, 3, 9, 5, 4, -1, -1, -1, -1, -1, -1, -1},
			{4, 9, 5, 0, 8, 1, 8, 10, 1, 8, 11, 10, -1, -1, -1, -1},
			{5, 4, 0, 5, 0, 11, 5, 11, 10, 11, 0, 3, -1, -1, -1, -1},
			{5, 4, 8, 5, 8, 10, 10, 8, 11, -1, -1, -1, -1, -1, -1, -1},
			{9, 7, 8, 5, 7, 9, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{9, 3, 0, 9, 5, 3, 5, 7, 3, -1, -1, -1, -1, -1, -1, -1},
			{0, 7, 8, 0, 1, 7, 1, 5, 7, -1, -1, -1, -1, -1, -1, -1},
			{1, 5, 3, 3, 5, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{9, 7, 8, 9, 5, 7, 10, 1, 2, -1, -1, -1, -1, -1, -1, -1},
			{10, 1, 2, 9, 5, 0, 5, 3, 0, 5, 7, 3, -1, -1, -1, -1},
			{8, 0, 2, 8, 2, 5, 8, 5, 7, 10, 5, 2, -1, -1, -1, -1},
			{2, 10, 5, 2, 5, 3, 3, 5, 7, -1, -1, -1, -1, -1, -1, -1},
			{7, 9, 5, 7, 8, 9, 3, 11, 2, -1, -1, -1, -1, -1, -1, -1},
			{9, 5, 7, 9, 7, 2, 9, 2, 0, 2, 7, 11, -1, -1, -1, -1},
			{2, 3, 11, 0, 1, 8, 1, 7, 8, 1, 5, 7, -1, -1, -1, -1},
			{11, 2, 1, 11, 1, 7, 7, 1, 5, -1, -1, -1, -1, -1, -1, -1},
			{9, 5, 8, 8, 5, 7, 10, 1, 3, 10, 3, 11, -1, -1, -1, -1},
			{5, 7, 0, 5, 0, 9, 7, 11, 0, 1, 0, 10, 11, 10, 0, -1},
			{11, 10, 0, 11, 0, 3, 10, 5, 0, 8, 0, 7, 5, 7, 0, -1},
			{11, 10, 5, 7, 11, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{10, 6, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{0, 8, 3, 5, 10, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{9, 0, 1, 5, 10, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{1, 8, 3, 1, 9, 8, 5, 10, 6, -1, -1, -1, -1, -1, -1, -1},
			{1, 6, 5, 2, 6, 1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{1, 6, 5, 1, 2, 6, 3, 0, 8, -1, -1, -1, -1, -1, -1, -1},
			{9, 6, 5, 9, 0, 6, 0, 2, 6, -1, -1, -1, -1, -1, -1, -1},
			{5, 9, 8, 5, 8, 2, 5, 2, 6, 3, 2, 8, -1, -1, -1, -1},
			{2, 3, 11, 10, 6, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{11, 0, 8, 11, 2, 0, 10, 6, 5, -1, -1, -1, -1, -1, -1, -1},
			{0, 1, 9, 2, 3, 11, 5, 10, 6, -1, -1, -1, -1, -1, -1, -1},
			{5, 10, 6, 1, 9, 2, 9, 11, 2, 9, 8, 11, -1, -1, -1, -1},
			{6, 3, 11, 6, 5, 3, 5, 1, 3, -1, -1, -1, -1, -1, -1, -1},
			{0, 8, 11, 0, 11, 5, 0, 5, 1, 5, 11, 6, -1, -1, -1, -1},
			{3, 11, 6, 0, 3, 6, 0, 6, 5, 0, 5, 9, -1, -1, -1, -1},
			{6, 5, 9, 6, 9, 11, 11, 9, 8, -1, -1, -1, -1, -1, -1, -1},
			{5, 10, 6, 4, 7, 8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{4, 3, 0, 4, 7, 3, 6, 5, 10, -1, -1, -1, -1, -1, -1, -1},
			{1, 9, 0, 5, 10, 6, 8, 4, 7, -1, -1, -1, -1, -1, -1, -1},
			{10, 6, 5, 1, 9, 7, 1, 7, 3, 7, 9, 4, -1, -1, -1, -1},
			{6, 1, 2, 6, 5, 1, 4, 7, 8, -1, -1, -1, -1, -1, -1, -1},
			{1, 2, 5, 5, 2, 6, 3, 0, 4, 3, 4, 7, -1, -1, -1, -1},
			{8, 4, 7, 9, 0, 5, 0, 6, 5, 0, 2, 6, -1, -1, -1, -1},
			{7, 3, 9, 7, 9, 4, 3, 2, 9, 5, 9, 6, 2, 6, 9, -1},
			{3, 11, 2, 7, 8, 4, 10, 6, 5, -1, -1, -1, -1, -1, -1, -1},
			{5, 10, 6, 4, 7, 2, 4, 2, 0, 2, 7, 11, -1, -1, -1, -1},
			{0, 1, 9, 4, 7, 8, 2, 3, 11, 5, 10, 6, -1, -1, -1, -1},
			{9, 2, 1, 9, 11, 2, 9, 4, 11, 7, 11, 4, 5, 10, 6, -1},
			{8, 4, 7, 3, 11, 5, 3, 5, 1, 5, 11, 6, -1, -1, -1, -1},
			{5, 1, 11, 5, 11, 6, 1, 0, 11, 7, 11, 4, 0, 4, 11, -1},
			{0, 5, 9, 0, 6, 5, 0, 3, 6, 11, 6, 3, 8, 4, 7, -1},
			{6, 5, 9, 6, 9, 11, 4, 7, 9, 7, 11, 9, -1, -1, -1, -1},
			{10, 4, 9, 6, 4, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{4, 10, 6, 4, 9, 10, 0, 8, 3, -1, -1, -1, -1, -1, -1, -1},
			{10, 0, 1, 10, 6, 0, 6, 4, 0, -1, -1, -1, -1, -1, -1, -1},
			{8, 3, 1, 8, 1, 6, 8, 6, 4, 6, 1, 10, -1, -1, -1, -1},
			{1, 4, 9, 1, 2, 4, 2, 6, 4, -1, -1, -1, -1, -1, -1, -1},
			{3, 0, 8, 1, 2, 9, 2, 4, 9, 2, 6, 4, -1, -1, -1, -1},
			{0, 2, 4, 4, 2, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{8, 3, 2, 8, 2, 4, 4, 2, 6, -1, -1, -1, -1, -1, -1, -1},
			{10, 4, 9, 10, 6, 4, 11, 2, 3, -1, -1, -1, -1, -1, -1, -1},
			{0, 8, 2, 2, 8, 11, 4, 9, 10, 4, 10, 6, -1, -1, -1, -1},
			{3, 11, 2, 0, 1, 6, 0, 6, 4, 6, 1, 10, -1, -1, -1, -1},
			{6, 4, 1, 6, 1, 10, 4, 8, 1, 2, 1, 11, 8, 11, 1, -1},
			{9, 6, 4, 9, 3, 6, 9, 1, 3, 11, 6, 3, -1, -1, -1, -1},
			{8, 11, 1, 8, 1, 0, 11, 6, 1, 9, 1, 4, 6, 4, 1, -1},
			{3, 11, 6, 3, 6, 0, 0, 6, 4, -1, -1, -1, -1, -1, -1, -1},
			{6, 4, 8, 11, 6, 8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{7, 10, 6, 7, 8, 10, 8, 9, 10, -1, -1, -1, -1, -1, -1, -1},
			{0, 7, 3, 0, 10, 7, 0, 9, 10, 6, 7, 10, -1, -1, -1, -1},
			{10, 6, 7, 1, 10, 7, 1, 7, 8, 1, 8, 0, -1, -1, -1, -1},
			{10, 6, 7, 10, 7, 1, 1, 7, 3, -1, -1, -1, -1, -1, -1, -1},
			{1, 2, 6, 1, 6, 8, 1, 8, 9, 8, 6, 7, -1, -1, -1, -1},
			{2, 6, 9, 2, 9, 1, 6, 7, 9, 0, 9, 3, 7, 3, 9, -1},
			{7, 8, 0, 7, 0, 6, 6, 0, 2, -1, -1, -1, -1, -1, -1, -1},
			{7, 3, 2, 6, 7, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{2, 3, 11, 10, 6, 8, 10, 8, 9, 8, 6, 7, -1, -1, -1, -1},
			{2, 0, 7, 2, 7, 11, 0, 9, 7, 6, 7, 10, 9, 10, 7, -1},
			{1, 8, 0, 1, 7, 8, 1, 10, 7, 6, 7, 10, 2, 3, 11, -1},
			{11, 2, 1, 11, 1, 7, 10, 6, 1, 6, 7, 1, -1, -1, -1, -1},
			{8, 9, 6, 8, 6, 7, 9, 1, 6, 11, 6, 3, 1, 3, 6, -1},
			{0, 9, 1, 11, 6, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{7, 8, 0, 7, 0, 6, 3, 11, 0, 11, 6, 0, -1, -1, -1, -1},
			{7, 11, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{7, 6, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{3, 0, 8, 11, 7, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{0, 1, 9, 11, 7, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{8, 1, 9, 8, 3, 1, 11, 7, 6, -1, -1, -1, -1, -1, -1, -1},
			{10, 1, 2, 6, 11, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{1, 2, 10, 3, 0, 8, 6, 11, 7, -1, -1, -1, -1, -1, -1, -1},
			{2, 9, 0, 2, 10, 9, 6, 11, 7, -1, -1, -1, -1, -1, -1, -1},
			{6, 11, 7, 2, 10, 3, 10, 8, 3, 10, 9, 8, -1, -1, -1, -1},
			{7, 2, 3, 6, 2, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{7, 0, 8, 7, 6, 0, 6, 2, 0, -1, -1, -1, -1, -1, -1, -1},
			{2, 7, 6, 2, 3, 7, 0, 1, 9, -1, -1, -1, -1, -1, -1, -1},
			{1, 6, 2, 1, 8, 6, 1, 9, 8, 8, 7, 6, -1, -1, -1, -1},
			{10, 7, 6, 10, 1, 7, 1, 3, 7, -1, -1, -1, -1, -1, -1, -1},
			{10, 7, 6, 1, 7, 10, 1, 8, 7, 1, 0, 8, -1, -1, -1, -1},
			{0, 3, 7, 0, 7, 10, 0, 10, 9, 6, 10, 7, -1, -1, -1, -1},
			{7, 6, 10, 7, 10, 8, 8, 10, 9, -1, -1, -1, -1, -1, -1, -1},
			{6, 8, 4, 11, 8, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{3, 6, 11, 3, 0, 6, 0, 4, 6, -1, -1, -1, -1, -1, -1, -1},
			{8, 6, 11, 8, 4, 6, 9, 0, 1, -1, -1, -1, -1, -1, -1, -1},
			{9, 4, 6, 9, 6, 3, 9, 3, 1, 11, 3, 6, -1, -1, -1, -1},
			{6, 8, 4, 6, 11, 8, 2, 10, 1, -1, -1, -1, -1, -1, -1, -1},
			{1, 2, 10, 3, 0, 11, 0, 6, 11, 0, 4, 6, -1, -1, -1, -1},
			{4, 11, 8, 4, 6, 11, 0, 2, 9, 2, 10, 9, -1, -1, -1, -1},
			{10, 9, 3, 10, 3, 2, 9, 4, 3, 11, 3, 6, 4, 6, 3, -1},
			{8, 2, 3, 8, 4, 2, 4, 6, 2, -1, -1, -1, -1, -1, -1, -1},
			{0, 4, 2, 4, 6, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{1, 9, 0, 2, 3, 4, 2, 4, 6, 4, 3, 8, -1, -1, -1, -1},
			{1, 9, 4, 1, 4, 2, 2, 4, 6, -1, -1, -1, -1, -1, -1, -1},
			{8, 1, 3, 8, 6, 1, 8, 4, 6, 6, 10, 1, -1, -1, -1, -1},
			{10, 1, 0, 10, 0, 6, 6, 0, 4, -1, -1, -1, -1, -1, -1, -1},
			{4, 6, 3, 4, 3, 8, 6, 10, 3, 0, 3, 9, 10, 9, 3, -1},
			{10, 9, 4, 6, 10, 4, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{4, 9, 5, 7, 6, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{0, 8, 3, 4, 9, 5, 11, 7, 6, -1, -1, -1, -1, -1, -1, -1},
			{5, 0, 1, 5, 4, 0, 7, 6, 11, -1, -1, -1, -1, -1, -1, -1},
			{11, 7, 6, 8, 3, 4, 3, 5, 4, 3, 1, 5, -1, -1, -1, -1},
			{9, 5, 4, 10, 1, 2, 7, 6, 11, -1, -1, -1, -1, -1, -1, -1},
			{6, 11, 7, 1, 2, 10, 0, 8, 3, 4, 9, 5, -1, -1, -1, -1},
			{7, 6, 11, 5, 4, 10, 4, 2, 10, 4, 0, 2, -1, -1, -1, -1},
			{3, 4, 8, 3, 5, 4, 3, 2, 5, 10, 5, 2, 11, 7, 6, -1},
			{7, 2, 3, 7, 6, 2, 5, 4, 9, -1, -1, -1, -1, -1, -1, -1},
			{9, 5, 4, 0, 8, 6, 0, 6, 2, 6, 8, 7, -1, -1, -1, -1},
			{3, 6, 2, 3, 7, 6, 1, 5, 0, 5, 4, 0, -1, -1, -1, -1},
			{6, 2, 8, 6, 8, 7, 2, 1, 8, 4, 8, 5, 1, 5, 8, -1},
			{9, 5, 4, 10, 1, 6, 1, 7, 6, 1, 3, 7, -1, -1, -1, -1},
			{1, 6, 10, 1, 7, 6, 1, 0, 7, 8, 7, 0, 9, 5, 4, -1},
			{4, 0, 10, 4, 10, 5, 0, 3, 10, 6, 10, 7, 3, 7, 10, -1},
			{7, 6, 10, 7, 10, 8, 5, 4, 10, 4, 8, 10, -1, -1, -1, -1},
			{6, 9, 5, 6, 11, 9, 11, 8, 9, -1, -1, -1, -1, -1, -1, -1},
			{3, 6, 11, 0, 6, 3, 0, 5, 6, 0, 9, 5, -1, -1, -1, -1},
			{0, 11, 8, 0, 5, 11, 0, 1, 5, 5, 6, 11, -1, -1, -1, -1},
			{6, 11, 3, 6, 3, 5, 5, 3, 1, -1, -1, -1, -1, -1, -1, -1},
			{1, 2, 10, 9, 5, 11, 9, 11, 8, 11, 5, 6, -1, -1, -1, -1},
			{0, 11, 3, 0, 6, 11, 0, 9, 6, 5, 6, 9, 1, 2, 10, -1},
			{11, 8, 5, 11, 5, 6, 8, 0, 5, 10, 5, 2, 0, 2, 5, -1},
			{6, 11, 3, 6, 3, 5, 2, 10, 3, 10, 5, 3, -1, -1, -1, -1},
			{5, 8, 9, 5, 2, 8, 5, 6, 2, 3, 8, 2, -1, -1, -1, -1},
			{9, 5, 6, 9, 6, 0, 0, 6, 2, -1, -1, -1, -1, -1, -1, -1},
			{1, 5, 8, 1, 8, 0, 5, 6, 8, 3, 8, 2, 6, 2, 8, -1},
			{1, 5, 6, 2, 1, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{1, 3, 6, 1, 6, 10, 3, 8, 6, 5, 6, 9, 8, 9, 6, -1},
			{10, 1, 0, 10, 0, 6, 9, 5, 0, 5, 6, 0, -1, -1, -1, -1},
			{0, 3, 8, 5, 6, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{10, 5, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{11, 5, 10, 7, 5, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{11, 5, 10, 11, 7, 5, 8, 3, 0, -1, -1, -1, -1, -1, -1, -1},
			{5, 11, 7, 5, 10, 11, 1, 9, 0, -1, -1, -1, -1, -1, -1, -1},
			{10, 7, 5, 10, 11, 7, 9, 8, 1, 8, 3, 1, -1, -1, -1, -1},
			{11, 1, 2, 11, 7, 1, 7, 5, 1, -1, -1, -1, -1, -1, -1, -1},
			{0, 8, 3, 1, 2, 7, 1, 7, 5, 7, 2, 11, -1, -1, -1, -1},
			{9, 7, 5, 9, 2, 7, 9, 0, 2, 2, 11, 7, -1, -1, -1, -1},
			{7, 5, 2, 7, 2, 11, 5, 9, 2, 3, 2, 8, 9, 8, 2, -1},
			{2, 5, 10, 2, 3, 5, 3, 7, 5, -1, -1, -1, -1, -1, -1, -1},
			{8, 2, 0, 8, 5, 2, 8, 7, 5, 10, 2, 5, -1, -1, -1, -1},
			{9, 0, 1, 5, 10, 3, 5, 3, 7, 3, 10, 2, -1, -1, -1, -1},
			{9, 8, 2, 9, 2, 1, 8, 7, 2, 10, 2, 5, 7, 5, 2, -1},
			{1, 3, 5, 3, 7, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{0, 8, 7, 0, 7, 1, 1, 7, 5, -1, -1, -1, -1, -1, -1, -1},
			{9, 0, 3, 9, 3, 5, 5, 3, 7, -1, -1, -1, -1, -1, -1, -1},
			{9, 8, 7, 5, 9, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{5, 8, 4, 5, 10, 8, 10, 11, 8, -1, -1, -1, -1, -1, -1, -1},
			{5, 0, 4, 5, 11, 0, 5, 10, 11, 11, 3, 0, -1, -1, -1, -1},
			{0, 1, 9, 8, 4, 10, 8, 10, 11, 10, 4, 5, -1, -1, -1, -1},
			{10, 11, 4, 10, 4, 5, 11, 3, 4, 9, 4, 1, 3, 1, 4, -1},
			{2, 5, 1, 2, 8, 5, 2, 11, 8, 4, 5, 8, -1, -1, -1, -1},
			{0, 4, 11, 0, 11, 3, 4, 5, 11, 2, 11, 1, 5, 1, 11, -1},
			{0, 2, 5, 0, 5, 9, 2, 11, 5, 4, 5, 8, 11, 8, 5, -1},
			{9, 4, 5, 2, 11, 3, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{2, 5, 10, 3, 5, 2, 3, 4, 5, 3, 8, 4, -1, -1, -1, -1},
			{5, 10, 2, 5, 2, 4, 4, 2, 0, -1, -1, -1, -1, -1, -1, -1},
			{3, 10, 2, 3, 5, 10, 3, 8, 5, 4, 5, 8, 0, 1, 9, -1},
			{5, 10, 2, 5, 2, 4, 1, 9, 2, 9, 4, 2, -1, -1, -1, -1},
			{8, 4, 5, 8, 5, 3, 3, 5, 1, -1, -1, -1, -1, -1, -1, -1},
			{0, 4, 5, 1, 0, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{8, 4, 5, 8, 5, 3, 9, 0, 5, 0, 3, 5, -1, -1, -1, -1},
			{9, 4, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{4, 11, 7, 4, 9, 11, 9, 10, 11, -1, -1, -1, -1, -1, -1, -1},
			{0, 8, 3, 4, 9, 7, 9, 11, 7, 9, 10, 11, -1, -1, -1, -1},
			{1, 10, 11, 1, 11, 4, 1, 4, 0, 7, 4, 11, -1, -1, -1, -1},
			{3, 1, 4, 3, 4, 8, 1, 10, 4, 7, 4, 11, 10, 11, 4, -1},
			{4, 11, 7, 9, 11, 4, 9, 2, 11, 9, 1, 2, -1, -1, -1, -1},
			{9, 7, 4, 9, 11, 7, 9, 1, 11, 2, 11, 1, 0, 8, 3, -1},
			{11, 7, 4, 11, 4, 2, 2, 4, 0, -1, -1, -1, -1, -1, -1, -1},
			{11, 7, 4, 11, 4, 2, 8, 3, 4, 3, 2, 4, -1, -1, -1, -1},
			{2, 9, 10, 2, 7, 9, 2, 3, 7, 7, 4, 9, -1, -1, -1, -1},
			{9, 10, 7, 9, 7, 4, 10, 2, 7, 8, 7, 0, 2, 0, 7, -1},
			{3, 7, 10, 3, 10, 2, 7, 4, 10, 1, 10, 0, 4, 0, 10, -1},
			{1, 10, 2, 8, 7, 4, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{4, 9, 1, 4, 1, 7, 7, 1, 3, -1, -1, -1, -1, -1, -1, -1},
			{4, 9, 1, 4, 1, 7, 0, 8, 1, 8, 7, 1, -1, -1, -1, -1},
			{4, 0, 3, 7, 4, 3, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{4, 8, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{9, 10, 8, 10, 11, 8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{3, 0, 9, 3, 9, 11, 11, 9, 10, -1, -1, -1, -1, -1, -1, -1},
			{0, 1, 10, 0, 10, 8, 8, 10, 11, -1, -1, -1, -1, -1, -1, -1},
			{3, 1, 10, 11, 3, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{1, 2, 11, 1, 11, 9, 9, 11, 8, -1, -1, -1, -1, -1, -1, -1},
			{3, 0, 9, 3, 9, 11, 1, 2, 9, 2, 11, 9, -1, -1, -1, -1},
			{0, 2, 11, 8, 0, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{3, 2, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{2, 3, 8, 2, 8, 10, 10, 8, 9, -1, -1, -1, -1, -1, -1, -1},
			{9, 10, 2, 0, 9, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{2, 3, 8, 2, 8, 10, 0, 1, 8, 1, 10, 8, -1, -1, -1, -1},
			{1, 10, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{1, 3, 8, 9, 1, 8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{0, 9, 1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{0, 3, 8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
			{-1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1}};

__global__
void PolygoniseCube(TRIANGLE * d_vectTriangles,GRIDCELL * g , int n ,int iso)
{
	//int iso=50;
	int ind = threadIdx.x + blockDim.x * blockIdx.x;
	//printf("este si %d \n",ind);		  
	//return ;
	if(ind<n)
	{
		//printf("thread %d \n", g[i].p[7].x);
		int cubeindex;
		//int tamVert=12;
		XYZ vertlist[12];
		cubeindex = 0;
		if (g[ind].val[0] < iso) cubeindex |= 1;
		if (g[ind].val[1] < iso) cubeindex |= 2;
		if (g[ind].val[2] < iso) cubeindex |= 4;
		if (g[ind].val[3] < iso) cubeindex |= 8;
		if (g[ind].val[4] < iso) cubeindex |= 16;
		if (g[ind].val[5] < iso) cubeindex |= 32;
		if (g[ind].val[6] < iso) cubeindex |= 64;
		if (g[ind].val[7] < iso) cubeindex |= 128;
		
		//printf("Est parte normal  %d \n",cubeindex );
		
			   if (edgeTable[cubeindex] == 0)
			      return;
			   /* Find the vertices where the surface intersects the cube */
			   if (edgeTable[cubeindex] & 1) {
			      vertlist[0] = VertexInterp(iso,g[ind].p[0],g[ind].p[1],g[ind].val[0],g[ind].val[1]);
			     // vertGradient[0]=  g[ind+1]; 
			   }
			   if (edgeTable[cubeindex] & 2) {
			      vertlist[1] = VertexInterp(iso,g[ind].p[1],g[ind].p[2],g[ind].val[1],g[ind].val[2]);
			   }
			   if (edgeTable[cubeindex] & 4) {
			      vertlist[2] = VertexInterp(iso,g[ind].p[2],g[ind].p[3],g[ind].val[2],g[ind].val[3]);
			   }
			   if (edgeTable[cubeindex] & 8) {
			      vertlist[3] = VertexInterp(iso,g[ind].p[3],g[ind].p[0],g[ind].val[3],g[ind].val[0]);
			   }
			   if (edgeTable[cubeindex] & 16) {
			      vertlist[4] = VertexInterp(iso,g[ind].p[4],g[ind].p[5],g[ind].val[4],g[ind].val[5]);
			   }
			   if (edgeTable[cubeindex] & 32) {
			      vertlist[5] = VertexInterp(iso,g[ind].p[5],g[ind].p[6],g[ind].val[5],g[ind].val[6]);
			   }
			   if (edgeTable[cubeindex] & 64) {
			      vertlist[6] = VertexInterp(iso,g[ind].p[6],g[ind].p[7],g[ind].val[6],g[ind].val[7]);
			   }
			   if (edgeTable[cubeindex] & 128) {
			      vertlist[7] = VertexInterp(iso,g[ind].p[7],g[ind].p[4],g[ind].val[7],g[ind].val[4]);
			   }
			   if (edgeTable[cubeindex] & 256) {
			      vertlist[8] = VertexInterp(iso,g[ind].p[0],g[ind].p[4],g[ind].val[0],g[ind].val[4]);
			   }
			   if (edgeTable[cubeindex] & 512) {
			      vertlist[9] = VertexInterp(iso,g[ind].p[1],g[ind].p[5],g[ind].val[1],g[ind].val[5]);
			   }
			   if (edgeTable[cubeindex] & 1024) {
			      vertlist[10] = VertexInterp(iso,g[ind].p[2],g[ind].p[6],g[ind].val[2],g[ind].val[6]);
			   }
			   if (edgeTable[cubeindex] & 2048) {
			      vertlist[11] = VertexInterp(iso,g[ind].p[3],g[ind].p[7],g[ind].val[3],g[ind].val[7]);
			   }
	   	int t=0; 
	   	//printf("indice %d \n", ind);	
	   	while(triTable[cubeindex][t] != -1)
	  	{
	      d_vectTriangles[ind].p[0] = vertlist[triTable[cubeindex][t]];
	      d_vectTriangles[ind].p[1] = vertlist[triTable[cubeindex][t+1]];
	      d_vectTriangles[ind].p[2] = vertlist[triTable[cubeindex][t+2]];
	      t+=3;
   		}
   		//printf("ntri %d \n", ntri);	
   }
}/*
void printGrid(string a, GRIDCELL * g, int tam)
{
	cout<<a;
	for(int i =0; i<tam ;i++)
		for(int j=0;j<8;j++)
			//printf("%f  %f  %f \n", g[i].p[j].x ,g[i].p[j].y,g[i].p[j].z);
		      printf("%f \n", g[i].val[j]);		
}
*/
int points( TRIANGLE * t,  int tam)
{
	int cont=0;
	for(int i =0; i<tam ;i++)
	{
		for (int k=0;k<3;k++)  
		{
			if(t[i].p[k].x != 0 && t[i].p[k].y != 0  && t[i].p[k].z !=0 )
			{
				cont++;	
			}
		}
	}
	return cont;
}

void printTriangles(string a , TRIANGLE * t,  int tam , int cont)
{
	cout<<a;
	//int cont=0;
	ofstream myfile;

	cout<<"-----------debe estar imprimiendo"<<endl;
  	myfile.open("Engine.pcd");
  	myfile<<"# .PCD v.5 - Point Cloud Data file format"<<endl;
	myfile<<"VERSION .5"<<endl;
	myfile<<"FIELDS x y z"<<endl;
	myfile<<"SIZE 4 4 4"<<endl;
	myfile<<"TYPE F F F"<<endl;
	myfile<<"COUNT 1 1 1"<<endl;
	myfile<<"WIDTH "<<cont<<endl;
	myfile<<"HEIGHT 1"<<endl;
	myfile<<"POINTS "<<cont<<endl;
	myfile<<"DATA ascii"<<endl;
	for(int i =0; i<tam ;i++)
	{
		for (int k=0;k<3;k++)  
		{
			//cont++;
			if(t[i].p[k].x != 0 && t[i].p[k].y != 0  && t[i].p[k].z !=0 )
			{
				myfile<<t[i].p[k].x<<" "<<t[i].p[k].y<<" "<<t[i].p[k].z<<endl;		
			}
		}
	}
	myfile.close();
}
int main(int argc, char *argv[])
{
	int i,j,k,c;
	int ***data;
	FILE *fptr;
	int NX =256;//200//200
	int NY =256;//160//160
	int NZ =256;//160//160
	int N= ((NX-1)*(NY-1)*(NZ-1));
	cout<<N<<endl; //return 1;
	int THREADS_PER_BLOCK =512;
	int themin=255;
	int themax=0;
	int isolevel=151;
	//const char* FILENAME = "mri.raw";
	//assingMem(data);
	//readFile(fptr,FILENAME,themin, themax,data);
	// Malloc the volumetric data, hardwired size!
	data = (int***)malloc(NX*sizeof(int **));
	for (i=0;i<NX;i++)
		data[i] = (int**)malloc(NY*sizeof(int *));
	for (i=0;i<NX;i++)
		for (j=0;j<NY;j++)
			data[i][j] = (int*)malloc(NZ*sizeof(int));
	
	//
	fprintf(stderr,"Leyendo Volumen ...\n");
	if ((fptr = fopen(argv[argc-1],"rb")) == NULL) {
		fprintf(stderr,"No se pudo abrir el archivo\n");
		exit(-1);
	}
	//cout<<"llega"<<endl;
	for (k=0;k<NZ;k++) {
		for (j=0;j<NY;j++) {
			for (i=0;i<NX;i++) {
				if ((c = fgetc(fptr)) == EOF) {
					fprintf(stderr,"Diferente dimesion de\n");
					exit(-1);
				}
				data[i][j][k] = c;
				//cout<<i<<" "<<j <<" "<<k <<" data : "<<data[i][j][k]<<endl;
				if (c > themax)
					themax = c;
				if (c < themin)
					themin = c;
			}
		}
	}
	fclose(fptr);

	long int sizeGRID= N*sizeof(GRIDCELL);
	long int sizeTRI = N*sizeof(TRIANGLE); 
	cout<<"sizeGRID "<<sizeGRID<<endl;
	cout<<"sizeTRI  "<<sizeTRI<<endl;
	GRIDCELL * vectGrids;
	GRIDCELL * d_vectGrids;
	TRIANGLE * vectTriangles;
	TRIANGLE * d_vectTriangles;

	vectGrids = (GRIDCELL *)malloc(sizeGRID);
	vectTriangles= (TRIANGLE *)malloc(sizeTRI); 


	cout<<"MEMORIA RAM"<<endl;
	cout<<"mem of grid "<<vectGrids<<endl;
	cout<<"mem of triangulos "<<vectTriangles<<endl;
	//cout<<"mem of normals"<<vectNormales<<endl;
	int numCubos = constructCubes(vectGrids,data,NX,NY,NZ);
	cout<<"pasa"<<endl;
		
	size_t available, total;
	hipMemGetInfo(&available, &total);
		cout<<"available:  " << available<<" total:  "<<total <<endl;
	hipMalloc((void **)&d_vectGrids, sizeGRID);
	hipMemcpy(d_vectGrids,vectGrids, sizeGRID, hipMemcpyHostToDevice);	
 
	 hipMemGetInfo(&available, &total);
		cout<<"available:  " << available<<" total:  "<<total <<endl;
 	hipMalloc((void **)&d_vectTriangles, sizeTRI);
	hipMemcpy(d_vectTriangles,vectTriangles,sizeTRI,hipMemcpyHostToDevice);

	hipMemGetInfo(&available, &total);
		cout<<"available:  " << available<<" total:  "<<total <<endl;
	cout<<"mem DEVICE of grid "<<d_vectGrids<<endl;
	cout<<"mem DEVICE of triangulos "<<d_vectTriangles<<endl;
	//cout<<"mem DEVICE of Normales "<<d_vectNormales<<endl;
	cout<<"ASIGNACION DEVICE NVIDIA"<<endl;
	//printGrid("imprimiendo Grid inicial en Host \n ",vectGrids,N);
	hipEvent_t start, stop;
	float elapsedTime;
	hipEventCreate(&start);
	int blocks= (N + THREADS_PER_BLOCK -1)/THREADS_PER_BLOCK;
	cout<<"blocks : \n"<<blocks<<"\n threds: \n "<<THREADS_PER_BLOCK<<endl; 

	hipEventRecord(start,0);
		PolygoniseCube<<<blocks,THREADS_PER_BLOCK>>>(d_vectTriangles,d_vectGrids,N,isolevel);
	hipEventCreate(&stop);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start,stop);
	printf("Tiempo utilizado: %f ms\n" ,elapsedTime);

	TRIANGLE * cpy_vectTriangles;
	cpy_vectTriangles= (TRIANGLE *)malloc(sizeTRI); 
	cout<<"crea sin problemas en host"<<endl;
	hipMemcpy(cpy_vectTriangles,d_vectTriangles, sizeTRI, hipMemcpyDeviceToHost);
	int NT= points(cpy_vectTriangles,N);
	printTriangles("Printing Tringles \n",cpy_vectTriangles,N,NT);
	system("pcl_viewer Engine.pcd");
	free(vectTriangles); free(vectGrids); free(cpy_vectTriangles);
	hipFree(d_vectTriangles); hipFree(d_vectGrids);
	return 0;
}
